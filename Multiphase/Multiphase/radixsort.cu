#include "hip/hip_runtime.h"
#ifndef _RADIXSORT_CU_
#define _RADIXSORT_CU_

#include "radixsort.cuh"
#include "radixsort_kernel.cu"
extern "C"
{

	////////////////////////////////////////////////////////////////////////////////
	//! Perform a radix sort
	//! Sorting performed in place on passed arrays.
	//!
	//! @param pData0       input and output array - data will be sorted
	//! @param pData1       additional array to allow ping pong computation
	//! @param elements     number of elements to sort
	////////////////////////////////////////////////////////////////////////////////
	void RadixSort(KeyValuePair *pData0, KeyValuePair *pData1, uint elements, uint bits)
	{
		// Round element count to total number of threads for efficiency
		uint elements_rounded_to_3072;
		int modval = elements % 3072;
		if (modval == 0)
			elements_rounded_to_3072 = elements;
		else
			elements_rounded_to_3072 = elements + (3072 - (modval));

		// Iterate over n bytes of y bit word, using each byte to sort the list in turn
		for (uint shift = 0; shift < bits; shift += RADIX)
		{
			// Perform one round of radix sorting

			// Generate per radix group sums radix counts across a radix group
			RadixSum << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, GRFSIZE >> >(pData0, elements, elements_rounded_to_3072, shift);
			// Prefix sum in radix groups, and then between groups throughout a block
			RadixPrefixSum << <PREFIX_NUM_BLOCKS, PREFIX_NUM_THREADS_PER_BLOCK, PREFIX_GRFSIZE >> >();
			// Sum the block offsets and then shuffle data into bins
			RadixAddOffsetsAndShuffle << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, SHUFFLE_GRFSIZE >> >(pData0, pData1, elements, elements_rounded_to_3072, shift);

			// Exchange data pointers
			KeyValuePair* pTemp = pData0;
			pData0 = pData1;
			pData1 = pTemp;
		}
	}

}//end of extern "C"


#endif
