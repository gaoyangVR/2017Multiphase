#include "hip/hip_runtime.h"
#include <Windows.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include"hip/hip_vector_types.h"
#include<helper_functions.h>
#include "glew.h"
#include<cuda_gl_interop.h>
#include "spray.h"
#include "spray_k.cuh"
#include "timer.h"
#include <hip/hip_runtime.h>
# include < hiprand.h >
#include "radixsort.cuh"

extern void printTime(bool btime, char* info, CTimer &time);

inline int getidx(int i, int j, int k)
{
	return (i*NZ*NY + j*NZ + k);
}


void cspray::initmem_bubble()
{
	copyparamtoGPU(hparam);
	copyNXNYNZtoGPU(NX, NY, NZ);

	//press
	int gsmemsize = sizeof(float)*hparam.gnum;
	hipMalloc((void**)&mpress.data, gsmemsize);
	hipMalloc((void**)&temppress.data, gsmemsize);
	hipMemset(mpress.data, 0, gsmemsize);
	hipMemset(temppress.data, 0, gsmemsize);

	//div
	hipMalloc((void**)&mDiv.data, gsmemsize);
	hipMemset(mDiv.data, 0, gsmemsize);

	//phi
	hipMalloc((void**)&phifluid.data, gsmemsize);
	hipMemset(phifluid.data, 0, gsmemsize);
	hipMalloc((void**)&phiair.data, gsmemsize);
	hipMemset(phiair.data, 0, gsmemsize);
	hipMalloc((void**)&phisolid.data, gsmemsize);
	hipMemset(phisolid.data, 0, gsmemsize);

	//level set value.
	hipMalloc((void**)&lsair.data, gsmemsize);
	hipMalloc((void**)&lsfluid.data, gsmemsize);
	hipMalloc((void**)&lsmerge.data, gsmemsize);
	//gradient of level set
	hipMalloc((void**)&phigrax.data, gsmemsize);
	hipMalloc((void**)&phigray.data, gsmemsize);
	hipMalloc((void**)&phigraz.data, gsmemsize);
	hipMalloc((void**)&phigrax_air.data, gsmemsize);
	hipMalloc((void**)&phigray_air.data, gsmemsize);
	hipMalloc((void**)&phigraz_air.data, gsmemsize);

	//surface tension
	hipMalloc((void**)&surfacetension.data, gsmemsize);

	//u
	int gvxmemsize = sizeof(float)*hparam.gvnum.x;
	int gvymemsize = sizeof(float)*hparam.gvnum.y;
	int gvzmemsize = sizeof(float)*hparam.gvnum.z;
	hipMalloc((void**)&waterux.data, gvxmemsize);
	waterux.setdim(NX + 1, NY, NZ);
	hipMalloc((void**)&wateruy.data, gvymemsize);
	wateruy.setdim(NX, NY + 1, NZ);
	hipMalloc((void**)&wateruz.data, gvzmemsize);
	wateruz.setdim(NX, NY, NZ + 1);

	hipMalloc((void**)&waterux_old.data, gvxmemsize);
	waterux_old.setdim(NX + 1, NY, NZ);
	hipMalloc((void**)&wateruy_old.data, gvymemsize);
	wateruy_old.setdim(NX, NY + 1, NZ);
	hipMalloc((void**)&wateruz_old.data, gvzmemsize);
	wateruz_old.setdim(NX, NY, NZ + 1);

	hipMalloc((void**)&tmpux.data, gvxmemsize);
	tmpux.setdim(NX + 1, NY, NZ);
	hipMalloc((void**)&tmpuy.data, gvymemsize);
	tmpuy.setdim(NX, NY + 1, NZ);
	hipMalloc((void**)&tmpuz.data, gvzmemsize);
	tmpuz.setdim(NX, NY, NZ + 1);

	hipMalloc((void**)&solidux.data, gvxmemsize);
	solidux.setdim(NX + 1, NY, NZ);
	hipMalloc((void**)&soliduy.data, gvymemsize);
	soliduy.setdim(NX, NY + 1, NZ);
	hipMalloc((void**)&soliduz.data, gvzmemsize);
	soliduz.setdim(NX, NY, NZ + 1);

	hipMemset(waterux.data, 0, gvxmemsize);
	hipMemset(wateruy.data, 0, gvymemsize);
	hipMemset(wateruz.data, 0, gvzmemsize);
	hipMemset(waterux_old.data, 0, gvxmemsize);
	hipMemset(wateruy_old.data, 0, gvymemsize);
	hipMemset(wateruz_old.data, 0, gvzmemsize);

	//for air u
	{
		hipMalloc((void**)&airux.data, gvxmemsize);
		airux.setdim(NX + 1, NY, NZ);
		hipMalloc((void**)&airuy.data, gvymemsize);
		airuy.setdim(NX, NY + 1, NZ);
		hipMalloc((void**)&airuz.data, gvzmemsize);
		airuz.setdim(NX, NY, NZ + 1);

		hipMalloc((void**)&airux_old.data, gvxmemsize);
		airux_old.setdim(NX + 1, NY, NZ);
		hipMalloc((void**)&airuy_old.data, gvymemsize);
		airuy_old.setdim(NX, NY + 1, NZ);
		hipMalloc((void**)&airuz_old.data, gvzmemsize);
		airuz_old.setdim(NX, NY, NZ + 1);

		hipMemset(airux.data, 0, gvxmemsize);
		hipMemset(airuy.data, 0, gvymemsize);
		hipMemset(airuz.data, 0, gvzmemsize);
		hipMemset(airux_old.data, 0, gvxmemsize);
		hipMemset(airuy_old.data, 0, gvymemsize);
		hipMemset(airuz_old.data, 0, gvzmemsize);
	}

	//mark
	hipMalloc((void**)&mmark, sizeof(char)*hparam.gnum);
	hipMalloc((void**)&mark_terrain, sizeof(char)*hparam.gnum);

	//particle
	hipMalloc((void**)&mParPos, parNumMax*sizeof(float3));
	hipMalloc((void**)&mParVel, parNumMax*sizeof(float3));
	hipMemset(mParVel, 0, parNumMax*sizeof(float3));
	hipMalloc((void**)&parflag, parNumMax*sizeof(char));
	hipMalloc((void**)&parmass, parNumMax*sizeof(float));
	hipMalloc((void**)&parTemperature, parNumMax*sizeof(float));
	hipMalloc((void**)&parLHeat, parNumMax*sizeof(float));
	hipMalloc((void**)&parsolubility, parNumMax*sizeof(float));
	hipMalloc((void**)&pargascontain, parNumMax*sizeof(float));
	//particle attribute: no need for sort
	hipMalloc((void**)&pardens, parNumMax*sizeof(float));
	hipMalloc((void**)&parpress, parNumMax*sizeof(float));

	//GY
	//hipMalloc((void**)&initialSolPos, parNumMax*sizeof(float3));/////////////////////////////GY
	hipMalloc((void**)&c, parNumMax*sizeof(float3));
	hipMalloc((void**)&I, parNumMax*sizeof(float3));
	hipMalloc((void**)&solidParPos, parNumMax*sizeof(float3));
	hipMalloc((void**)&solidParVelFLIP, parNumMax*sizeof(float3));
	// 	hipMemset( c, 0, parNumMax*sizeof(float3));
	// 	hipMemset( I, 0, parNumMax*sizeof(float3));

	//for deleting particles.
	hipMalloc((void**)&preservemark, parNumMax*sizeof(uint));
	hipMalloc((void**)&preservemarkscan, parNumMax*sizeof(uint));

	//sort the particles.
	hipMalloc((void**)&gridHash, parNumMax*sizeof(uint));
	hipMalloc((void**)&gridIndex, parNumMax*sizeof(uint));
	hipMalloc((void**)&gridstart, 8 * hparam.gnum*sizeof(uint));
	hipMalloc((void**)&gridend, 8 * hparam.gnum*sizeof(uint));
	hipMalloc((void**)&tmpParPos, parNumMax*sizeof(float3));
	hipMalloc((void**)&tmpParVelFLIP, parNumMax*sizeof(float3));
	hipMalloc((void**)&tmpparflag, parNumMax*sizeof(char));
	hipMalloc((void**)&tmpparmass, parNumMax*sizeof(float));
	hipMalloc((void**)&tmpparTemperature, parNumMax*sizeof(float));
	hipMalloc((void**)&tmpparHeat, parNumMax*sizeof(float));
	hipMalloc((void**)&tempsolubility, parNumMax*sizeof(float));
	hipMalloc((void**)&tempgascontain, parNumMax*sizeof(float));



	//for pcg
	hipMalloc((void**)&pre.data, sizeof(float)*hparam.gnum);
	hipMalloc((void**)&z.data, sizeof(float)*hparam.gnum);
	hipMalloc((void**)&r.data, sizeof(float)*hparam.gnum);
	hipMalloc((void**)&p.data, sizeof(float)*hparam.gnum);

	//for rand number of CUDA
	{
		hipMalloc((void**)&randfloat, sizeof(float)*randfloatcnt);
		hiprandGenerator_t gen;
		// Create pseudo - random number generator 
		(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
		/* Set seed */
		(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
		//����0.0~1.0f�������
		(hiprandGenerateUniform(gen, randfloat, randfloatcnt));
	}

	//�ȴ������¶�
	hipMalloc((void**)&Tp.data, sizeof(float)*hparam.gnum);
	hipMalloc((void**)&Tp_old.data, sizeof(float)*hparam.gnum);
	hipMalloc((void**)&Tp_save.data, sizeof(float)*hparam.gnum);
	hipMalloc((void**)&fixedHeat.data, sizeof(float)*hparam.gnum);

	//seed
	hipMalloc((void**)&dseedcell, sizeof(int)*seednum);

	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//for CPU computation.
	hparLHeat = new float[parNumMax];
	hparflag = new char[parNumMax];
	hwaterux.setdim(NX + 1, NY, NZ);
	hwateruy.setdim(NX, NY + 1, NZ);
	hwateruz.setdim(NX, NY, NZ + 1);
	hwaterux.data = new float[hparam.gvnum.x];
	hwateruy.data = new float[hparam.gvnum.y];
	hwateruz.data = new float[hparam.gvnum.z];
	hpos = new float3[parNumMax];
	hvel = new float3[parNumMax];
	hmass = new float[parNumMax];
	hgridstart = new uint[hparam.gnum];
	hgridend = new uint[hparam.gnum];
}

float cspray::checkGridFarray(farray u)
{
	static float *ux2 = new float[gvblocknum*threadnum];
	hipMemcpy(ux2, u.data, sizeof(float)*u.xn*u.yn*u.zn, hipMemcpyDeviceToHost);

	float sum = 0.0f;
	for (int idx = 0; idx<u.xn*u.yn*u.zn; idx++)
	{
		int i, j, k;
		getijk(i, j, k, idx, u.xn, u.yn, u.zn);
		if (!(ux2[idx]>-1 || ux2[idx]<1))
		{
			printf("i,j,k=%d,%d,%d, value=%f\n", i, j, k, ux2[idx]);

			break;
		}
		//sum+=abs(ux2[idx]);
	}
	return sum;
}

void cspray::project_CG(farray ux, farray uy, farray uz)
{
	CTimer time;
	time.startTimer();
	hipMemset(mDiv.data, 0, sizeof(float)*hparam.gnum);
	cptdivergence << <gsblocknum, threadnum >> >(mDiv, ux, uy, uz, mmark);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//solve pressure
	hipMemset(mpress.data, 0, sizeof(float)*hparam.gnum);

	solver_cg(mmark, mpress, mDiv, hparam.gnum);

	// 	int cnt=0;
	// 	static char* hmark=new char[hparam.gnum];
	// 	hipMemcpy( hmark, mmark.data, sizeof(char)*hparam.gnum, hipMemcpyDeviceToHost);
	// 	for( int i=0; i<hparam.gnum; ++i )
	// 		if( hmark[i]==TYPEFLUID )
	// 			cnt++;
	// 	printf( "typefluid=%d\n", cnt );
	// 
	// 	float tsum = checkGridFarray(mpress);
	// 	printf( "sum of press = %f\n", tsum );

	//compute divergence-free velocity.
	subGradPress << <gvblocknum, threadnum >> >(mpress, ux, uy, uz);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}


void cspray::project_Jacobi(farray ux, farray uy, farray uz)
{
	CTimer time;
	time.startTimer();
	hipMemset(mDiv.data, 0, sizeof(float)*hparam.gnum);
	cptdivergence << <gsblocknum, threadnum >> >(mDiv, ux, uy, uz, mmark);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//solve pressure
	hipMemset(mpress.data, 0, sizeof(float)*hparam.gnum);

	solver_Jacobi(mmark, mpress, mDiv, MAXITER);

	//compute divergence-free velocity.
	subGradPress << <gvblocknum, threadnum >> >(mpress, ux, uy, uz);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::wateradvect()
{
#if 0
	advectparticle << <pblocknum, threadnum >> >(mParPos, mParVel, parNumNow,
		mwaterux, mwateruy, mwateruz, hparam.dt, parflag, velmode);
#else

	advectparticle_RK2 << <pblocknum, threadnum >> >(mParPos, mParVel, parNumNow,
		waterux, wateruy, wateruz, hparam.dt, parflag, velmode);
#endif
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::copyParticle2GL()
{
	float* renderPos, *rendercolor;
	cudaGLMapBufferObject((void**)&renderPos, vboParPos);
	cudaGLMapBufferObject((void**)&rendercolor, vboParColor);

	printf("rendering, parnumnow=%d\n", parNumNow);
	if (simmode == SIMULATION_SOLIDCOUPLING)
		copyParticle2GL_vel_k << <pblocknum, threadnum >> >(mParPos, mParVel, parmass, parflag, parNumNow, renderPos, rendercolor);
	else
		copyParticle2GL_phi << <pblocknum, threadnum >> >(mParPos, parflag, parmass, parTemperature, parNumNow, renderPos, rendercolor,
		lsmerge, phigrax, phigray, phigraz, renderpartiletype, temperatureMax_render, temperatureMin_render);

	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	cudaGLUnmapBufferObject(vboParPos);
	cudaGLUnmapBufferObject(vboParColor);
}

//�ο���https://code.google.com/p/flip3d/
void cspray::correctpos()
{
	correctparticlepos << <pblocknum, threadnum >> >(tmpParPos, mParPos, parmass, parflag, parNumNow, gridstart, gridend, correctionspring, correctionradius,
		pEmptyPos, pEmptyRadius, pEmptyNum);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	float3 *temp = tmpParPos;
	tmpParPos = mParPos;
	mParPos = temp;
}

void cspray::mapvelp2g()
{
	mapvelp2g_k_fluidSolid << <gvblocknum, threadnum >> >(mParPos, mParVel, parmass, parflag, parNumNow, waterux, wateruy, wateruz, gridstart, gridend);
	//mapvelp2g_slow<<<gvblocknum, threadnum>>>(mParPos, mParVel, mParNum, mUx, mUy, mUz );
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	if (velmode == FLIP)
	{
		hipMemcpy(waterux_old.data, waterux.data, sizeof(float)*hparam.gvnum.x, hipMemcpyDeviceToDevice);
		hipMemcpy(wateruy_old.data, wateruy.data, sizeof(float)*hparam.gvnum.y, hipMemcpyDeviceToDevice);
		hipMemcpy(wateruz_old.data, wateruz.data, sizeof(float)*hparam.gvnum.z, hipMemcpyDeviceToDevice);
	}
}

inline void swappointer(farray &a, farray &b)
{
	float* temp = a.data;
	a.data = b.data;
	b.data = temp;
}

void cspray::mapvelg2p()
{
	if (velmode == FLIP)	//in CIP mode, vel of partciles will be updated in advect part.
	{		//ע�⣬��Ҫ��һ���ٶȵĸı䷽ʽ����delta_v������v
		computeDeltaU << <gvblocknum, threadnum >> >(waterux, wateruy, wateruz, waterux_old, wateruy_old, wateruz_old);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");

		mapvelg2p_flip << <pblocknum, threadnum >> >(mParPos, mParVel, parflag, parNumNow, waterux_old, wateruy_old, wateruz_old);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");
	}
}

void cspray::addexternalforce()
{
	// 	if( mscene==SCENE_BUBBLE || mscene==SCENE_MULTIBUBBLE )
	// 		addgravityforce_k<<<pblocknum, threadnum>>>(mParVelFLIP, parflag, parTemperature, parNumNow, hparam.dt, Temperature0 );
	// 	else
	addgravityforce_k << <pblocknum, threadnum >> >(mParVel, parflag, parNumNow, hparam.dt);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//addbuoyancyforce_k<<<pblocknum, threadnum>>>( buoyantHeight, mParPos,mParVel, parflag, parNumNow, hparam.dt );

	addbuoyancyforce_vel << <pblocknum, threadnum >> >(bubbleMaxVel, mParPos, mParVel, parflag, parNumNow, hparam.dt, buoyanceRateAir, buoyanceRateSolo);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	// 	buoyancyForSolid<<<pblocknum, threadnum>>>( mParPos, mParVel, parflag, parNumNow, gridstart, gridend, SolidBuoyanceParam );
	// 	hipDeviceSynchronize();
}

void cspray::initparticle_solidCoupling()
{
	if (parNumNow <= 0)
		return;

	float3* hparpos = new float3[parNumNow];//����λ��
	float3* hparvel = new float3[parNumNow];//�����ٶ�
	float* hparmass = new float[parNumNow];//��������
	char* hparflag = new char[parNumNow];	//���ӱ��
	float x, y, z;

	int i = 0, ParNumPerLevel = 0;
	for (float z = hparam.cellsize.x + hparam.samplespace; z<0.8f * NZ*hparam.cellsize.x && i + ParNumPerLevel<initfluidparticle; z += hparam.samplespace)
	{
		// 		for( float y = hparam.cellsize.x+hparam.samplespace; y<hparam.cellsize.x*(0.7*NY-1)-0.5f*hparam.samplespace && i<initfluidparticle; y+=hparam.samplespace )
		// 			for( float x = hparam.cellsize.x+hparam.samplespace; x<hparam.cellsize.x*(0.7*NX-1)-0.5f*hparam.samplespace && i<initfluidparticle; x+=hparam.samplespace )
		// 			{
		for (float y = hparam.cellsize.x + hparam.samplespace; y<hparam.cellsize.x*(NY - 1) - 0.5f*hparam.samplespace && i<initfluidparticle; y += hparam.samplespace)
		for (float x = hparam.cellsize.x + hparam.samplespace; x<hparam.cellsize.x*(NX - 1) - 0.5f*hparam.samplespace && i<initfluidparticle; x += hparam.samplespace)
		{
			hparpos[i] = make_float3(x, y, z);
			hparvel[i] = make_float3(0.0f);
			hparmass[i] = hparam.m0;
			hparflag[i] = TYPEFLUID;
			++i;
		}
		if (ParNumPerLevel == 0) ParNumPerLevel = i;
	}

	float scale = 50;
	if (mscene == SCENE_FREEZING || mscene == SCENE_MELTINGPOUR) scale = 80;
	if (mscene == SCENE_INTERACTION) scale = 60;
	if (mscene == SCENE_MELTANDBOIL_HIGHRES || mscene == SCENE_INTERACTION_HIGHRES) scale = 100;

	if (m_bSolid)
	{
		for (int j = 0; j<nInitSolPoint; j++)
		{
			x = float(SolpointPos[j][0]), y = float(SolpointPos[j][1]), z = float(SolpointPos[j][2]);
			hparpos[i] = hparam.samplespace*make_float3(x, y, z)*scale + solidInitPos;
			hparvel[i] = make_float3(0.0f);		//	
			hparmass[i] = hparam.m0*0.8f;
			hparflag[i] = TYPESOLID;	//�����ǹ���

			++i;
		}
	}
	parNumNow = i;

	hipMemcpy(mParPos, hparpos, sizeof(float3)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(mParVel, hparvel, sizeof(float3)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(parmass, hparmass, sizeof(float)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(parflag, hparflag, sizeof(char)*parNumNow, hipMemcpyHostToDevice);

	delete[] hparpos;
	delete[] hparvel;
	delete[] hparmass;
	delete[] hparflag;
}

void cspray::markgrid()
{
	markair << <gsblocknum, threadnum >> >(mmark);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//markfluid<<<pblocknum, threadnum>>>( mmark, mParPos, parflag, parNumNow );
	//todo: ������������⣡����
	//markfluid_GY<<<pblocknum, threadnum>>>( mmark, mParPos, parflag, parNumNow );
	markfluid << <pblocknum, threadnum >> >(mmark, mParPos, parflag, parNumNow);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	if (bCouplingSphere)
	{
		markSolid_sphere << <gsblocknum, threadnum >> >(solidInitPos, sphereradius, mmark);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");
	}

	//todo: ������������⣡����
	markBoundaryCell << <gsblocknum, threadnum >> >(mmark);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::flipMark_sphere()
{
	flipAirVacuum << <gsblocknum, threadnum >> >(mmark);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::markgrid_bubble()
{
	int fluidParCntPerGridThres = 10;
	markfluid_dense << <pblocknum, threadnum >> >(mmark, parmass, parflag, parNumNow, gridstart, gridend, fluidParCntPerGridThres);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
	markBoundaryCell << <gsblocknum, threadnum >> >(mmark);
}

void cspray::smokemarkgrid()
{
	markforsmoke << <gsblocknum, threadnum >> >(mmark, spraydense);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	if (bCouplingSphere)
	{
		markSolid_sphere << <gsblocknum, threadnum >> >(solidInitPos, sphereradius, mmark);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");
	}

	markBoundaryCell << <gsblocknum, threadnum >> >(mmark);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::PrintMemInfo()
{
	size_t freeMem, totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
	printf("freeMem=%fM,totalMem=%fM\n", freeMem / 1024.0f / 1024.0f, totalMem / 1024.0f / 1024.0f);
}

void cspray::setGridColor()
{
	float* color;
	cudaGLMapBufferObject((void**)&color, vboGridcolor);

	//printf( "%d\n", (int)colormode);
	setgridcolor_k << <gsblocknum, threadnum >> >(
		color, colormode, mpress, waterux_old, wateruy_old, wateruz_old, mDiv, phiair, mmark, lsmerge, Tp, surfacetensionsigma, temperatureMax_render, temperatureMin_render);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	cudaGLUnmapBufferObject(vboGridcolor);
}

void cspray::sweepPhi(farray phi, char typeflag)
{
	initphi << <gsblocknum, threadnum >> >(phi, mmark, typeflag);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	for (int it = 0; it<3; it++)
	{
		sweepphibytype << <gsblocknum, threadnum >> >(phi, mmark, typeflag);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");
	}
}

void cspray::sweepU(farray ux, farray uy, farray uz, farray phi, charray mark, char typeflag)
{
	for (int it = 0; it<2; it++)
	{
		//sweepu<<<gvblocknum, threadnum>>>( tmpux,tmpuy,tmpuz, ux, uy, uz, phi, mark );
		sweepu_k_bubble << <gvblocknum, threadnum >> >(tmpux, tmpuy, tmpuz, ux, uy, uz, phi, mark, typeflag);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");
		swappointer(tmpux, ux);
		swappointer(tmpuy, uy);
		swappointer(tmpuz, uz);
	}
}

void cspray::setSmokeBoundaryU(farray ux, farray uy, farray uz)
{
	setSmokeBoundaryU_k << <gvblocknum, threadnum >> >(ux, uy, uz, mmark);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::setWaterBoundaryU(farray ux, farray uy, farray uz)
{
	setWaterBoundaryU_k << <gvblocknum, threadnum >> >(ux, uy, uz, mmark);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

extern void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles);

void cspray::hashAndSortParticles()
{
	calcHashD << <pblocknum, threadnum >> >(gridHash, gridIndex, mParPos, parNumNow);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	sortParticles(gridHash, gridIndex, parNumNow);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//��Ҫ��һ������������
	swapParticlePointers();

	uint smemSize = sizeof(uint)*(threadnum + 1);
	(hipMemset(gridstart, CELL_UNDEF, hparam.gnum*sizeof(uint)));
	reorderDataAndFindCellStartD << <pblocknum, threadnum, smemSize >> >(
		gridstart, gridend,
		mParPos, mParVel, parflag, parmass, parTemperature, parLHeat, parsolubility, pargascontain,
		gridHash, gridIndex,
		tmpParPos, tmpParVelFLIP, tmpparflag, tmpparmass, tmpparTemperature, tmpparHeat, tempsolubility, tempgascontain,
		parNumNow);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::hashAndSortParticles_MC()
{
	calcHashD_MC << <pblocknum, threadnum >> >(gridHash, gridIndex, mParPos, parNumNow);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	sortParticles(gridHash, gridIndex, parNumNow);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//��Ҫ��һ������������
	swapParticlePointers();

	uint smemSize = sizeof(uint)*(threadnum + 1);
	(hipMemset(gridstart, CELL_UNDEF, NXMC*NYMC*NZMC*sizeof(uint)));
	reorderDataAndFindCellStartD << <pblocknum, threadnum, smemSize >> >(
		gridstart, gridend,
		mParPos, mParVel, parflag, parmass, parTemperature, parLHeat, parsolubility, pargascontain,
		gridHash, gridIndex,
		tmpParPos, tmpParVelFLIP, tmpparflag, tmpparmass, tmpparTemperature, tmpparHeat, tempsolubility, tempgascontain,
		parNumNow);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::smokeadvection()
{
	//advect concertration by the div-free velocity field.
	advectscaler << <gsblocknum, threadnum >> >(tmpspraydense, spraydense, msprayux, msprayuy, msprayuz, densedissipation, wind);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//advect velocity field by itself.
	advectux << <gvblocknum, threadnum >> >(mtmpsprayux, msprayux, msprayuy, msprayuz, velocitydissipation, wind);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
	advectuy << <gvblocknum, threadnum >> >(mtmpsprayuy, msprayux, msprayuy, msprayuz, velocitydissipation, wind);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
	advectuz << <gvblocknum, threadnum >> >(mtmpsprayuz, msprayux, msprayuy, msprayuz, velocitydissipation, wind);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	swappointer(msprayux, mtmpsprayux);
	swappointer(msprayuy, mtmpsprayuy);
	swappointer(msprayuz, mtmpsprayuz);
	swappointer(tmpspraydense, spraydense);
}

void cspray::smokesetvel()
{
	setsmokedense << <gsblocknum, threadnum >> > (spraydense);
	hipDeviceSynchronize();
	setsmokevel << <gvblocknum, threadnum >> >(msprayuz, spraydense);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::copyDensity2GL()
{
	hipGraphicsMapResources(1, &densTex3D_cuda, 0);
	hipArray *cudaarray;
	hipGraphicsSubResourceGetMappedArray(&cudaarray, densTex3D_cuda, 0, 0);
	writedens2surface(cudaarray, gsblocknum, threadnum, tmpspraydense);

	hipGraphicsUnmapResources(1, &densTex3D_cuda, 0);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::initMC()
{
	printf("Before MC: "), PrintMemInfo();

	uint* d_numVertsTable = 0;
	uint* d_edgeTable = 0;
	uint* d_triTable = 0;

	allocateTextures(&d_edgeTable, &d_triTable, &d_numVertsTable);

	int rate = 2 * 2 * 2;
	numVoxels = hparam.gnum * rate;		//NOTICE: *8 for double resolution
	maxVerts = 4000000;// hparam.gnum*1*rate;
	maxTriangles = 8000000;//hparam.gnum*2*rate;

	centertmp = make_float3(0);

	// allocate device memory
	unsigned int memSize = sizeof(uint)* numVoxels;
	checkCudaErrors(hipMalloc((void**)&d_voxelVerts, memSize));
	checkCudaErrors(hipMalloc((void**)&d_voxelVertsScan, memSize));
	checkCudaErrors(hipMalloc((void**)&d_voxelOccupied, memSize));
	checkCudaErrors(hipMalloc((void**)&d_voxelOccupiedScan, memSize));
	checkCudaErrors(hipMalloc((void**)&d_compVoxelArray, memSize));

	MCedgeNum = 3 * (NX + 1)*(NY + 1)*(NZ + 1) * rate;			//NOTICE: *8 for double resolution
	hipMalloc((void**)&MCedgemark, MCedgeNum*sizeof(uint));
	hipMalloc((void**)&MCedgemarkScan, MCedgeNum*sizeof(uint));

	initMCtriangles(maxVerts, maxTriangles);
	//for smoothing the triangles in 3d mesh created by MC
	hipMalloc((void**)&smoothdisplacement, maxVerts*sizeof(float3));
	hipMalloc((void**)&smoothweight, maxVerts*sizeof(int));
	hipMemset(smoothdisplacement, 0, maxVerts*sizeof(float3));		//necessary!!!
	hipMemset(smoothweight, 0, maxVerts*sizeof(int));		//necessary!!!

	maxsolidvert = 300000, maxsolidtri = 600000;
	hipMalloc((void**)&solidvertex, maxsolidvert*sizeof(float3));
	hipMalloc((void**)&solidnormal, maxsolidvert*sizeof(float3));
	hipMalloc((void**)&solidindices, maxsolidtri * 3 * sizeof(uint));

	//for marching cube.
	hipMalloc((void**)&waterdensMC.data, sizeof(float)*(NX + 1)*(NY + 1)*(NZ + 1)*rate);
	waterdensMC.setdim(NX + 1, NY + 1, NZ + 1);

	printf("After MC: "), PrintMemInfo();
}

extern void ThrustScanWrapper(unsigned int* output, unsigned int* input, unsigned int numElements);

void cspray::preMC()
{
	NXMC = NX * 2, NYMC = NY * 2, NZMC = NZ * 2;
	copyNXNYNZtoGPU_MC(NXMC, NYMC, NZMC);

	hashAndSortParticles_MC();
}

void cspray::runMC_fluid()
{
	//��TYPEAIR, TYPEAIRSOLO��TYPEFLUID������MC��������
	m_bLiquidAndGas = true;
	runMC_smooth("water", TYPEFLUID);
	m_bLiquidAndGas = false;
}

void cspray::runMC_solid()
{
	runMC_smooth("solid", TYPESOLID);
}

void cspray::runMC_gas()
{
	//��TYPEAIR, TYPEAIRSOLO������MC��������
	m_bGas = true;
	runMC_smooth("gas", TYPEAIR);
	m_bGas = false;
}

void cspray::runMC_interaction()
{
	if (mframe>0 && !bRunMCSolid)//just output, they are updated in solidmotion function.
	{
		calNormals(solidnormal, solidvertex, solidvertexnum, solidindices, solidindicesnum);

		//output
		if (boutputpovray && mframe%outputframeDelta == 0)
			outputPovRaywater(mframe / outputframeDelta, solidvertex, solidnormal, solidvertexnum, solidindices, solidindicesnum, "solid");
	}
	else
		runMC_smooth("solid", TYPESOLID);
}

void cspray::runMC_smooth(const char* objectname, char MCParType)
{
	//1. gen the density field
	int blocknum = (int)ceil(((float)(NXMC + 1)*(NYMC + 1)*(NZMC + 1)) / threadnum);
	waterdensMC.setdim(NXMC + 1, NYMC + 1, NZMC + 1);

	//todo: gridstart and gridend.

	if (m_bLiquidAndGas)	// MC both liquid and gas particle.
		genWaterDensfield_liquidAndGas << <blocknum, threadnum >> >(waterdensMC, mParPos, parflag, gridstart, gridend, fMCDensity);
	else if (m_bGas)	// MC both liquid and gas particle.
		genWaterDensfield_Gas << <blocknum, threadnum >> >(waterdensMC, mParPos, parflag, gridstart, gridend, fMCDensity, mscene);
	else	if (m_DistanceFuncMC == 0)
	{
		genWaterDensfield_GY << <blocknum, threadnum >> >(waterdensMC, mParPos, parflag, gridstart, gridend, fMCDensity, MCParType, centertmp);
		//printf("dis_GY\n");
	}
	else
	{
		genWaterDensfield2 << <blocknum, threadnum >> >(waterdensMC, mParPos, parflag, gridstart, gridend, fMCDensity, MCParType);
		//printf("dis2\n");
	}

	//test
	// 	float3 pos = make_float3(0.25f, 0.25f, 0.25f + (mframe%100) * 0.001f );
	// 	genSphereDensfield<<<blocknum, threadnum>>>(waterdensMC, pos, 0.07f );

	//2. calculate number of vertices need per voxel
	blocknum = (int)ceil(((float)(NXMC)*(NYMC)*(NZMC)) / threadnum);
	numVoxels = NXMC*NYMC*NZMC;
	classifyVoxel << <blocknum, threadnum >> >(d_voxelVerts, d_voxelOccupied, waterdensMC, 0);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//3. scan voxel occupied array
	ThrustScanWrapper(d_voxelOccupiedScan, d_voxelOccupied, numVoxels);

	// read back values to calculate total number of non-empty voxels
	// since we are using an exclusive scan, the total is the last value of
	// the scan result plus the last value in the input array
	{
		uint lastElement, lastScanElement;
		checkCudaErrors(hipMemcpy((void *)&lastElement,
			(void *)(d_voxelOccupied + numVoxels - 1),
			sizeof(uint), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy((void *)&lastScanElement,
			(void *)(d_voxelOccupiedScan + numVoxels - 1),
			sizeof(uint), hipMemcpyDeviceToHost));
		activeVoxels = lastElement + lastScanElement;
	}

	if (activeVoxels == 0) {
		// return if there are no full voxels
		totalVerts = 0;

		if (boutputpovray && mframe%outputframeDelta == 0)
			outputPovRaywater(mframe / outputframeDelta, NULL, NULL, 0, NULL, 0, objectname);
		return;
	}

	//4. compact voxel index array
	compactVoxels << <blocknum, threadnum >> >(d_compVoxelArray, d_voxelOccupied,
		d_voxelOccupiedScan, numVoxels);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//5. scan voxel totalTriagles count array
	ThrustScanWrapper(d_voxelVertsScan, d_voxelVerts, numVoxels);
	// readback total number of totalTriagles
	{
		uint lastElement, lastScanElement;
		checkCudaErrors(hipMemcpy((void *)&lastElement,
			(void *)(d_voxelVerts + numVoxels - 1),
			sizeof(uint), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy((void *)&lastScanElement,
			(void *)(d_voxelVertsScan + numVoxels - 1),
			sizeof(uint), hipMemcpyDeviceToHost));
		totalIndices = (lastElement + lastScanElement);
		//		printf("indices number = %u\n", totalIndices);

		if (totalIndices>maxTriangles * 3)
		{
			printf("MC totalIndices exceeds, ERROR!!\n");
			mpause = true;
			return;
		}
	}

	//6. generate triangles, writing to vertex buffers
	{
		size_t num_bytes;
		float3 *d_pos, *d_normal;
		uint *d_indices;		//todo: gl memory
		checkCudaErrors(hipGraphicsMapResources(1, &res_posvbo, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_pos, &num_bytes, res_posvbo));
		checkCudaErrors(hipGraphicsMapResources(1, &res_normvbo, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_normal, &num_bytes, res_normvbo));
		checkCudaErrors(hipGraphicsMapResources(1, &res_indicesvbo, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_indices, &num_bytes, res_indicesvbo));

		dim3 grid2((int)ceil(activeVoxels / (float)NTHREADS), 1, 1);
		while (grid2.x > 65535) {
			grid2.x /= 2;
			grid2.y *= 2;
		}

		//7. begin��Ϊ�˰��������ε�������ϵ����Ҫ���⴦��
		{
			hipMemset(MCedgemark, 0, MCedgeNum*sizeof(uint));

			markActiveEdge_MC << <grid2, NTHREADS >> >(MCedgemark, d_compVoxelArray, waterdensMC, 0, activeVoxels);
			hipDeviceSynchronize();
			getLastCudaError("Kernel execution failed");
			// scan voxel vertex count array

			ThrustScanWrapper(MCedgemarkScan, MCedgemark, MCedgeNum);

			// readback total number of vertices
			{
				uint lastElement, lastScanElement;
				checkCudaErrors(hipMemcpy((void *)&lastElement,
					(void *)(MCedgemark + MCedgeNum - 1),
					sizeof(uint), hipMemcpyDeviceToHost));
				checkCudaErrors(hipMemcpy((void *)&lastScanElement,
					(void *)(MCedgemarkScan + MCedgeNum - 1),
					sizeof(uint), hipMemcpyDeviceToHost));
				totalVerts = lastElement + lastScanElement;
			}
			if (totalVerts>maxVerts)
			{
				printf("MC total verts exceed, ERROR!!\n");
				mpause = true;
				return;
			}
			//	printf("totalVerts number = %u\n", totalVerts);

			generateTriangles_indices << <grid2, NTHREADS >> >(d_pos, d_indices, d_compVoxelArray, waterdensMC, 0,
				activeVoxels, maxVerts, MCedgemarkScan, d_voxelVertsScan);
			hipDeviceSynchronize();
			getLastCudaError("Kernel execution failed");
		}

		//8. smooth and calculate normals of vertices.
		if (m_bSmoothMC)
			smoothMesh(d_pos, totalVerts, d_indices, totalIndices / 3);
		calNormals(d_normal, d_pos, totalVerts, d_indices, totalIndices);

		//output
		if (boutputpovray && mframe%outputframeDelta == 0)
			outputPovRaywater(mframe / outputframeDelta, d_pos, d_normal, totalVerts, d_indices, totalIndices, objectname);

		if ((mscene == SCENE_INTERACTION || mscene == SCENE_INTERACTION_HIGHRES || mscene == SCENE_MELTANDBOIL || mscene == SCENE_MELTANDBOIL_HIGHRES) && objectname == "solid")
		{
			if (totalVerts>maxsolidvert || totalIndices>maxsolidtri * 3)		//mem is not enough, error.
			{
				printf("MC for solid: vert and triangle are too many!!! ERROR!\n");
				mpause = true;
				return;
			}
			hipMemcpy(solidvertex, d_pos, totalVerts*sizeof(float3), hipMemcpyDeviceToDevice);
			hipMemcpy(solidindices, d_indices, totalIndices*sizeof(uint), hipMemcpyDeviceToDevice);
			solidvertexnum = totalVerts;
			solidindicesnum = totalIndices;
		}

		printf(" vert num=%d, indices=%d\n", totalVerts, totalIndices);

		checkCudaErrors(hipGraphicsUnmapResources(1, &res_normvbo, 0));
		checkCudaErrors(hipGraphicsUnmapResources(1, &res_posvbo, 0));
		checkCudaErrors(hipGraphicsUnmapResources(1, &res_indicesvbo, 0));
	}
}

void cspray::calNormals(float3 *dnormals, float3 *dpos, int vertexnum, uint *dindices, int indicesnum)
{
	//set to 0
	hipMemset(dnormals, 0, vertexnum * 3 * sizeof(float));

	//calculate face normal
	int faceblocknum = max(1, (int)ceil(indicesnum / 3.0f / threadnum));
	calnormal_k << <faceblocknum, threadnum >> >(dpos, dnormals, vertexnum, dindices, indicesnum);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//calculate vertex normal.
	int vertexblocknum = max(1, (int)ceil(((float)vertexnum) / threadnum));
	normalizeTriangleNor_k << <vertexblocknum, threadnum >> >(dnormals, vertexnum);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

bool verifyfloat3(float3 &a)
{
	if (!(a.x>0 || a.x <1))
		return false;
	if (!(a.y>0 || a.y <1))
		return false;
	if (!(a.z>0 || a.z <1))
		return false;
	return true;
}

void cspray::runMC_flat(char MCParType)
{
	//gen the density field
	int blocknum = (int)ceil(((float)(NX + 1)*(NY + 1)*(NZ + 1)) / threadnum);

	//	genWaterDensfield<<<blocknum, threadnum>>>( waterdensMC, mParPos, parflag, gridstart, gridend, fMCDensity);
	//genSphereDensfield<<<blocknum, threadnum>>>( waterdensMC );
	genWaterDensfield_GY << <blocknum, threadnum >> >(waterdensMC, mParPos, parflag, gridstart, gridend, fMCDensity, MCParType, centertmp);

	// calculate number of vertices need per voxel
	classifyVoxel << <gsblocknum, threadnum >> >(d_voxelVerts, d_voxelOccupied, waterdensMC, 0);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	// scan voxel occupied array
	ThrustScanWrapper(d_voxelOccupiedScan, d_voxelOccupied, numVoxels);

	// read back values to calculate total number of non-empty voxels
	// since we are using an exclusive scan, the total is the last value of
	// the scan result plus the last value in the input array
	{
		uint lastElement, lastScanElement;
		checkCudaErrors(hipMemcpy((void *)&lastElement,
			(void *)(d_voxelOccupied + hparam.gnum - 1),
			sizeof(uint), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy((void *)&lastScanElement,
			(void *)(d_voxelOccupiedScan + hparam.gnum - 1),
			sizeof(uint), hipMemcpyDeviceToHost));
		activeVoxels = lastElement + lastScanElement;
	}

	if (activeVoxels == 0) {
		// return if there are no full voxels
		totalVerts = 0;
		return;
	}

	// compact voxel index array
	compactVoxels << <gsblocknum, threadnum >> >(d_compVoxelArray, d_voxelOccupied,
		d_voxelOccupiedScan, hparam.gnum);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	// scan voxel vertex count array
	ThrustScanWrapper(d_voxelVertsScan, d_voxelVerts, numVoxels);
	// readback total number of vertices
	{
		uint lastElement, lastScanElement;
		checkCudaErrors(hipMemcpy((void *)&lastElement,
			(void *)(d_voxelVerts + numVoxels - 1),
			sizeof(uint), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy((void *)&lastScanElement,
			(void *)(d_voxelVertsScan + numVoxels - 1),
			sizeof(uint), hipMemcpyDeviceToHost));
		totalVerts = lastElement + lastScanElement;
	}
	printf("totalVerts number = %u\n", totalVerts);

	// generate triangles, writing to vertex buffers
	{
		size_t num_bytes;
		float3 *d_pos, *d_normal;
		// DEPRECATED: checkCudaErrors(cudaGLMapBufferObject((void**)&d_pos, posVbo));
		checkCudaErrors(hipGraphicsMapResources(1, &res_posvbo, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_pos, &num_bytes, res_posvbo));

		// DEPRECATED: checkCudaErrors(cudaGLMapBufferObject((void**)&d_normal, normalVbo));
		checkCudaErrors(hipGraphicsMapResources(1, &res_normvbo, 0));
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&d_normal, &num_bytes, res_normvbo));

		dim3 grid2((int)ceil(activeVoxels / (float)NTHREADS), 1, 1);
		while (grid2.x > 65535) {
			grid2.x /= 2;
			grid2.y *= 2;
		}
		generateTriangles2 << <grid2, NTHREADS >> >(d_pos, d_normal, d_compVoxelArray, d_voxelVertsScan, waterdensMC, 0,
			activeVoxels, maxVerts);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");
		// DEPRECATED: 		checkCudaErrors(cudaGLUnmapBufferObject(normalVbo));
		checkCudaErrors(hipGraphicsUnmapResources(1, &res_normvbo, 0));
		// DEPRECATED: 		checkCudaErrors(cudaGLUnmapBufferObject(posVbo));
		checkCudaErrors(hipGraphicsUnmapResources(1, &res_posvbo, 0));
	}
}

int cspray::getblocknum(int n)
{
	return (int)ceil(((float)n) / threadnum);
}

float cspray::product(farray a, farray b, int n)
{
	static float *dsum = NULL;
	if (!dsum)
		hipMalloc((void**)&dsum, sizeof(float)*gsblocknum);
	static float *hsum = new float[gsblocknum];
	int sharememsize = threadnum*sizeof(float);

	arrayproduct_k << <gsblocknum, threadnum, sharememsize >> >(dsum, a.data, b.data, n);
	hipDeviceSynchronize();
	hipMemcpy(hsum, dsum, sizeof(float)*gsblocknum, hipMemcpyDeviceToHost);
	float res = 0;
	for (int i = 0; i<gsblocknum; i++)
		res += hsum[i];//, printf( "debug: hsum = %f\n", hsum[i]);
	return res;
}

bool verifyfloat(float a)
{
	if (!(a>0 || a<1))
		return false;
	return true;
}

bool cspray::solver_cg(charray A, farray x, farray b, int n)
{
	//todo: build precondition

	float a, zp, error2, eps = ((float)(1e-2))*n, alpha, beta;
	// z = applyA(x)
	computeAx << <gsblocknum, threadnum >> >(z, A, x, n);
	// r = b-Ax = b-1.0*z
	pcg_op << <gsblocknum, threadnum >> >(A, r, b, z, -1.0, n);

	error2 = product(r, r, n);
	if (error2<eps)
		return true;
	//error2 = product( r, r, n );	//error2 = r * r

	//todo: applyPreconditioner
	//buildprecondition_pcg<<<gsblocknum,threadnum>>>( pre, A, z,r,n);
	hipMemcpy(z.data, r.data, sizeof(float)*n, hipMemcpyDeviceToDevice);	//z=r;

	hipMemcpy(p.data, z.data, sizeof(float)*n, hipMemcpyDeviceToDevice);	//p=z;

	a = product(z, r, n);	//a = z*r;

	int k = 0;
	for (; k<n; k++)
	{
		//for debug.
		if (k >= 800 && k % 100 == 0)
		{
			if (!verifyfloat(a) || !verifyfloat(alpha) || !verifyfloat(error2))
				printf("there is some illegal float number in PCG solver!!!\n");
			printf("pcg iteration times: %d\n", k);
		}

		//z=A*p
		computeAx << <gsblocknum, threadnum >> >(z, A, p, n);

		zp = product(z, p, n);	//zp = z*p
		if (zp == 0)
			return true;

		alpha = a / zp;		//alpha = a/(z . p) = z*r/(z*p)

		// x = x + alpha*p
		pcg_op << <gsblocknum, threadnum >> >(A, x, x, p, alpha, n);
		// r = r - alpha*z;
		pcg_op << <gsblocknum, threadnum >> >(A, r, r, z, -alpha, n);

		//error2 = r * r
		error2 = product(r, r, n);
		if (error2<eps)
			break;

		//todo: applyPreconditioner
		hipMemcpy(z.data, r.data, sizeof(float)*n, hipMemcpyDeviceToDevice);	//z=r;
		//buildprecondition_pcg<<<gsblocknum,threadnum>>>( pre, A, z,r,n);

		//a2 = z*r;
		float a2 = product(z, r, n);

		beta = a2 / a;                     // beta = a2 / a
		// p = z + beta*p
		pcg_op << <gsblocknum, threadnum >> >(A, p, z, p, beta, n);

		a = a2;
	}
	// 	if( mtime )
	// 		printf("CG interation: %d\n", k );

	return true;
}

void cspray::solver_Jacobi(charray A, farray x, farray b, int itertime)
{
	hipMemset(temppress.data, 0, sizeof(float)*hparam.gnum);
	for (int i = 0; i<itertime / 2; i++)
	{
		JacobiIter << <gsblocknum, threadnum >> >(temppress, x, b, A);
		hipDeviceSynchronize();

		JacobiIter << <gsblocknum, threadnum >> >(x, temppress, b, A);
		hipDeviceSynchronize();
	}
}

void cspray::swapParticlePointers()
{
	float3* temp;
	temp = mParPos, mParPos = tmpParPos, tmpParPos = temp;
	temp = mParVel, mParVel = tmpParVelFLIP, tmpParVelFLIP = temp;
	float* temp2;
	temp2 = tmpparmass, tmpparmass = parmass, parmass = temp2;
	temp2 = tmpparTemperature, tmpparTemperature = parTemperature, parTemperature = temp2;
	temp2 = tmpparHeat, tmpparHeat = parLHeat, parLHeat = temp2;
	temp2 = tempsolubility, tempsolubility = parsolubility, parsolubility = temp2;
	temp2 = tempgascontain, tempgascontain = pargascontain, pargascontain = temp2;
	char* temp3;
	temp3 = tmpparflag, tmpparflag = parflag, parflag = temp3;
}

void cspray::checkdensesum()
{
	static float *hdense = new float[hparam.gnum];
	hipMemcpy(hdense, spraydense.data, sizeof(float)*hparam.gnum, hipMemcpyDeviceToHost);
	float sum = 0;
	for (int i = 0; i<hparam.gnum; i++)
		sum += hdense[i];
	printf("dense sum = %f\n", sum);

	static float dense0 = sum;
	printf("dense0 = %f\n", dense0);
	for (int i = 0; i<hparam.gnum; i++)
		hdense[i] *= dense0 / sum;
	hipMemcpy(spraydense.data, hdense, sizeof(float)*hparam.gnum, hipMemcpyHostToDevice);
}

void cspray::checkparticlevariables(float3* dvel)
{
	//debug if off.
#if 1
	static float3* hvel = new float3[parNumMax];
	hipMemcpy(hvel, dvel, parNumNow*sizeof(float3), hipMemcpyDeviceToHost);
	for (int i = 0; i<parNumNow; i++)
	{
		if (!(hvel[i].x>-1 || hvel[i].x<1))
		{
			printf("i=%d, particle velocity x=%f!!!!!\n", i, hvel[i].x);
			mpause = true;
			return;
		}
		if (!(hvel[i].y>-1 || hvel[i].y<1))
		{
			printf("i=%d, particle velocity y=%f!!!!!\n", i, hvel[i].y);
			mpause = true;
			return;
		}
		if (!(hvel[i].z>-1 || hvel[i].z<1))
		{
			printf("i=%d, particle velocity z=%f!!!!!\n", i, hvel[i].z);
			mpause = true;
			return;
		}
	}
#endif
}

void cspray::smoothMesh(float3 *dpos, int vertexnum, uint *indices, int trianglenum)
{
	int fblocknum = max(1, (int)ceil(((float)trianglenum) / threadnum));
	int vblocknum = max(1, (int)ceil(((float)vertexnum) / threadnum));
	float lambda = 0.5f;
	float mu = -0.53f;

	for (int i = 0; i<smoothIterTimes; i++)
	{
		smooth_computedisplacement << <fblocknum, threadnum >> >(smoothdisplacement, smoothweight, dpos, indices, trianglenum);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");
		smooth_addDisplacement << <vblocknum, threadnum >> >(smoothdisplacement, smoothweight, dpos, vertexnum, lambda);
		hipDeviceSynchronize();
		smooth_addDisplacement << <vblocknum, threadnum >> >(smoothdisplacement, smoothweight, dpos, vertexnum, mu);
		hipDeviceSynchronize();
	}
}

//[GPU Gems]Fast Fluid Dynamics Simulation on the GPU
void cspray::smokediffuse()
{
	//diffuse dense field.
	float alpha, beta;
	alpha = hparam.cellsize.x*hparam.cellsize.x / hparam.dt / fDenseDiffuse;
	beta = 6 + alpha;
	for (int i = 0; i<nDiffuseIters / 2; i++)
	{
		diffuse_dense << <gsblocknum, threadnum >> >(tmpspraydense, spraydense, mmark, alpha, beta);
		hipDeviceSynchronize();
		diffuse_dense << <gsblocknum, threadnum >> >(spraydense, tmpspraydense, mmark, alpha, beta);
		hipDeviceSynchronize();
	}
}

void cspray::ComputeTriangleHashSize(myMesh &mesh)
{
	//1. �ѳ����е�С������hash����
	int numFaces = mesh.m_nFaces;
	int nGridDim = (int)ceil(((float)numFaces) / threadnum);

	float* dMaxLength, *hHashSize;
	checkCudaErrors(hipMalloc((void**)&dMaxLength, sizeof(float)*nGridDim));
	hHashSize = new float[nGridDim];

	createAABB_q << <nGridDim, threadnum >> >(mesh.m_dPoints,
		mesh.m_nPoints, mesh.m_dFaces, numFaces, dMaxLength, mesh.m_dHashPointsForFaces);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//2. ����������Hash����Ĵ�С 
	hipMemcpy(hHashSize, dMaxLength, sizeof(float)*nGridDim, hipMemcpyDeviceToHost);
	float hashSize = hHashSize[0];
	for (int i = 1; i<nGridDim; i++)
	{
		hashSize = max(hHashSize[i], hashSize);
	}

	checkCudaErrors(hipFree(dMaxLength));
	delete[] hHashSize;

	hashSize = hashSize / 2 + hparam.pradius;

	hparam.triHashSize = make_float3(hashSize);
	hparam.triHashRes.x = ceil((mesh.m_max.x - mesh.m_min.x) / hparam.triHashSize.x);
	hparam.triHashRes.y = ceil((mesh.m_max.y - mesh.m_min.y) / hparam.triHashSize.y);
	hparam.triHashRes.z = ceil((mesh.m_max.z - mesh.m_min.z) / hparam.triHashSize.z);
	hparam.triHashCells = (int)(hparam.triHashRes.x*hparam.triHashRes.y*hparam.triHashRes.z);

	copyparamtoGPU(hparam);

	//����ռ�
	checkCudaErrors(hipMalloc((void**)&mesh.m_dTriCellStart, sizeof(uint)*hparam.triHashCells));
	checkCudaErrors(hipMalloc((void**)&mesh.m_dTriCellEnd, sizeof(uint)*hparam.triHashCells));
}

void cspray::hashTriangle_radix_q()
{
	int numFaces = mmesh.m_nFaces;
	int nGridDim = (int)ceil(((float)numFaces) / threadnum);

	calcHash_radix_q << < nGridDim, threadnum >> >((uint2*)mmesh.m_dTriHash_radix[0], mmesh.m_dHashPointsForFaces, numFaces, mmesh.m_min, mmesh.m_max);

	getLastCudaError("Kernel execution failed");
	hipDeviceSynchronize();
}

void cspray::sortTriangles_q(uint numParticles)
{
	RadixSort((KeyValuePair *)mmesh.m_dTriHash_radix[0], (KeyValuePair *)mmesh.m_dTriHash_radix[1], numParticles, 32);
	getLastCudaError("Kernel execution failed");
	hipDeviceSynchronize();
}


void cspray::reorderTriangle_radix_q()
{
	int numFaces = mmesh.m_nFaces;

	checkCudaErrors(hipMemset(mmesh.m_dTriCellStart, CELL_UNDEF, hparam.triHashCells * sizeof(uint)));
	int nGridDim = (int)ceil(((float)numFaces) / threadnum);
	uint smemSize = sizeof(uint)*(threadnum + 1);


	reorderDataAndFindCellStart_radix_q << <nGridDim, threadnum, smemSize >> >(
		mmesh.m_dTriCellStart, mmesh.m_dTriCellEnd,
		mmesh.m_dFacesSorted,
		(uint2*)mmesh.m_dTriHash_radix[0],
		mmesh.m_dFaces,
		numFaces);


	getLastCudaError("Kernel execution failed");
	hipDeviceSynchronize();

}
void cspray::updateNormal_q()
{

	int numFaces = mmesh.m_nFaces;
	int nGridDim = (int)ceil(((float)numFaces) / threadnum);

	calculateNormal << <nGridDim, threadnum >> >(mmesh.m_dPoints, mmesh.m_dFacesSorted, mmesh.m_dFaceNormals, mmesh.m_nFaces);

	getLastCudaError("Kernel execution failed");
	hipDeviceSynchronize();
}

void cspray::initscene_bubble()
{
	if (parNumNow <= 0)
		return;

	float3* hparpos = new float3[parNumNow];
	float3* hparvel = new float3[parNumNow];
	float* hparmass = new float[parNumNow];
	//	float* htemperature = new float[parNumNow];
	char* hparflag = new char[parNumNow];

	float3 bubblepos = make_float3(NX*0.5f, NY*0.5f, 8.5f) * hparam.cellsize.x;		//bottom
	float bubbleradius2 = hparam.cellsize.x * 0.6f;		//small bubble

	//1. ��ʼ�����岿��
	int i = 0;
	for (float z = hparam.cellsize.x + hparam.samplespace; z<0.8f * NZ*hparam.cellsize.x && i<initfluidparticle; z += hparam.samplespace)
	{
		for (float y = hparam.cellsize.x + hparam.samplespace; y<hparam.cellsize.x*(NY - 1) - 0.5f*hparam.samplespace && i<initfluidparticle; y += hparam.samplespace)
		for (float x = hparam.cellsize.x + hparam.samplespace; x<hparam.cellsize.x*(NX - 1) - 0.5f*hparam.samplespace && i<initfluidparticle; x += hparam.samplespace)
		{
			hparpos[i] = make_float3(x, y, z);
			hparvel[i] = make_float3(0.0f);
			hparmass[i] = hparam.m0;

			if (length(hparpos[i] - bubblepos)<bubbleradius2)
				hparflag[i] = TYPEAIR;
			else
				hparflag[i] = TYPEFLUID;

			++i;
		}
	}
	parNumNow = i;
	printf("init fluid/air particle succeed, parnum=%d.\n", parNumNow);

	//2. ����й���Ļ�����ʼ�����岿��
	if (m_bSolid)
	{
		for (int j = 0; j<nInitSolPoint; j++)
		{
			float x = float(SolpointPos[j][0]), y = float(SolpointPos[j][1]), z = float(SolpointPos[j][2]);
			hparpos[i + j] = hparam.samplespace*make_float3(x, y, z) * 50 + solidInitPos;
			//	printf( "%f,%f,%f\n", hparpos[i+j].x, hparpos[i+j].y, hparpos[i+j].z );

			hparvel[i + j] = make_float3(0.0f);		//	
			hparmass[i + j] = hparam.m0*0.8f;
			hparflag[i + j] = TYPESOLID;	//�����ǹ���6692963
		}
		parNumNow += nInitSolPoint;
		printf("init solid particle succeed, parnum=%d.\n", parNumNow);
	}

	hipMemcpy(mParPos, hparpos, sizeof(float3)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(mParVel, hparvel, sizeof(float3)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(parmass, hparmass, sizeof(float)*parNumNow, hipMemcpyHostToDevice);
	//	hipMemcpy( parTemperature, htemperature, sizeof(float)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(parflag, hparflag, sizeof(char)*parNumNow, hipMemcpyHostToDevice);

	delete[] hparpos;
	delete[] hparvel;
	delete[] hparmass;
	//delete [] htemperature;
	delete[] hparflag;
}

void cspray::computeLevelset(float offset)
{
	//ע�����������mark��levelset�Ǻܹؼ��ģ���С�ĶԴ���
	genlevelset << <gsblocknum, threadnum >> >(lsfluid, lsair, mmark, mParPos, parflag, parmass, gridstart, gridend, fMCDensity, offset);
	getLastCudaError("Kernel execution failed");
	hipDeviceSynchronize();
}

inline bool verifycellidx(int i, int j, int k)
{
	if (i<0 || i>NX - 1 || j<0 || j>NY - 1 || k<0 || k>NZ - 1)
		return false;
	return true;
}

inline float sharp_kernel(float r2, float h)
{
	return fmax(h*h / fmax(r2, 0.0001f) - 1.0f, 0.0f);
}

void sumcell_host(float3 &usum, float &weight, float3 gpos, float3 *pos, float3 *vel, float *mass, uint *gridstart, uint  *gridend, int gidx)
{
	if (gridstart[gidx] == CELL_UNDEF)
		return;
	uint start = gridstart[gidx];
	uint end = gridend[gidx];
	float dis2, w, RE = 1.4;
	float scale = 64;
	for (uint p = start; p<end; ++p)
	{
		dis2 = dot(pos[p] * scale - gpos, pos[p] * scale - gpos);		//scale is necessary.
		w = mass[p] * sharp_kernel(dis2, RE);
		weight += w;
		usum += w*vel[p];
	}
}

void cspray::mapvelp2g_bubble_CPU()
{
	hipMemcpy(hgridstart, gridstart, hparam.gnum*sizeof(uint), hipMemcpyDeviceToHost);
	hipMemcpy(hgridend, gridend, hparam.gnum*sizeof(uint), hipMemcpyDeviceToHost);
	hipMemcpy(hpos, mParPos, parNumNow*sizeof(float3), hipMemcpyDeviceToHost);
	hipMemcpy(hvel, mParVel, parNumNow*sizeof(float3), hipMemcpyDeviceToHost);
	hipMemcpy(hmass, parmass, parNumNow*sizeof(float), hipMemcpyDeviceToHost);

	CTimer time;
	time.startTimer();

	int vnum = max((NX + 1)*NY*NZ, (NX)*(NY + 1)*NZ);
	vnum = max(vnum, NX*NY*(NZ + 1));
	float weight;
	float3 gpos, usum;
	int i, j, k;
	for (int idx = 0; idx<vnum; idx++)
	{
		//ux
		if (idx<hparam.gvnum.x)
		{
			weight = 0, usum = make_float3(0.0f);
			getijk(i, j, k, idx, NX + 1, NY, NZ);
			gpos.x = i, gpos.y = j + 0.5, gpos.z = k + 0.5;
			for (int di = -1; di <= 0; di++) for (int dj = -1; dj <= 1; dj++) for (int dk = -1; dk <= 1; dk++)
			if (verifycellidx(i + di, j + dj, k + dk))
				sumcell_host(usum, weight, gpos, hpos, hvel, hmass, hgridstart, hgridend, getidx(i + di, j + dj, k + dk));

			usum.x = (weight>0) ? (usum.x / weight) : 0.0f;
			hwaterux(i, j, k) = usum.x;
		}
		// uy
		if (idx<hparam.gvnum.y)
		{
			weight = 0, usum = make_float3(0.0f);
			getijk(i, j, k, idx, NX, NY + 1, NZ);
			gpos.x = i + 0.5, gpos.y = j, gpos.z = k + 0.5;
			for (int di = -1; di <= 1; di++) for (int dj = -1; dj <= 0; dj++) for (int dk = -1; dk <= 1; dk++)
			if (verifycellidx(i + di, j + dj, k + dk))
				sumcell_host(usum, weight, gpos, hpos, hvel, hmass, hgridstart, hgridend, getidx(i + di, j + dj, k + dk));
			usum.y = (weight>0) ? (usum.y / weight) : 0.0f;
			hwateruy(i, j, k) = usum.y;
		}
		// uz
		if (idx<hparam.gvnum.z)
		{
			weight = 0, usum = make_float3(0.0f);
			getijk(i, j, k, idx, NX, NY, NZ + 1);
			gpos.x = i + 0.5, gpos.y = j + 0.5, gpos.z = k;
			for (int di = -1; di <= 1; di++) for (int dj = -1; dj <= 1; dj++) for (int dk = -1; dk <= 0; dk++)
			if (verifycellidx(i + di, j + dj, k + dk))
				sumcell_host(usum, weight, gpos, hpos, hvel, hmass, hgridstart, hgridend, getidx(i + di, j + dj, k + dk));
			usum.z = (weight>0) ? (usum.z / weight) : 0.0f;
			hwateruz(i, j, k) = usum.z;
		}

	}
	printTime(m_bCPURun, "mapvelp2g_bubble_CPU", time);
}

void cspray::mapvelp2g_bubble()
{

	if (m_bCPURun)
		mapvelp2g_bubble_CPU();

	CTimer time;
	time.startTimer();
	mapvelp2g_k_fluidSolid << <gvblocknum, threadnum >> >(mParPos, mParVel, parmass, parflag, parNumNow, waterux, wateruy, wateruz, gridstart, gridend);
	hipDeviceSynchronize();
	printTime(m_bCPURun, "mapvelp2g_k_fluidSolid", time);

	mapvelp2g_k_air << <gvblocknum, threadnum >> >(mParPos, mParVel, parmass, parflag, parNumNow, airux, airuy, airuz, gridstart, gridend);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	if (velmode == FLIP)
	{
		hipMemcpy(waterux_old.data, waterux.data, sizeof(float)*hparam.gvnum.x, hipMemcpyDeviceToDevice);
		hipMemcpy(wateruy_old.data, wateruy.data, sizeof(float)*hparam.gvnum.y, hipMemcpyDeviceToDevice);
		hipMemcpy(wateruz_old.data, wateruz.data, sizeof(float)*hparam.gvnum.z, hipMemcpyDeviceToDevice);

		hipMemcpy(airux_old.data, airux.data, sizeof(float)*hparam.gvnum.x, hipMemcpyDeviceToDevice);
		hipMemcpy(airuy_old.data, airuy.data, sizeof(float)*hparam.gvnum.y, hipMemcpyDeviceToDevice);
		hipMemcpy(airuz_old.data, airuz.data, sizeof(float)*hparam.gvnum.z, hipMemcpyDeviceToDevice);
	}
}

void cspray::project_CG_bubble()
{
	CTimer time;
	time.startTimer();
	hipMemset(mDiv.data, 0, sizeof(float)*hparam.gnum);
	cptdivergence_bubble2 << <gsblocknum, threadnum >> >(mDiv, waterux, wateruy, wateruz, airux, airuy, airuz, mmark, lsmerge);
	//cptdivergence<<<gsblocknum,threadnum>>>( mDiv, waterux, wateruy, wateruz, mmark );
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//solve pressure
	hipMemset(mpress.data, 0, sizeof(float)*hparam.gnum);
	solver_cg_bubble(mmark, mpress, mDiv, hparam.gnum);

	//compute divergence-free velocity.
	subGradPress << <gvblocknum, threadnum >> >(mpress, waterux, wateruy, wateruz);
	subGradPress << <gvblocknum, threadnum >> >(mpress, airux, airuy, airuz);
	// subGradPress_bubble<<<gvblocknum,threadnum>>>(mpress, waterux, wateruy, wateruz, surfacetension, lsmerge, mmark );
	//	subGradPress_bubble<<<gvblocknum,threadnum>>>(mpress, airux, airuy, airuz, surfacetension, lsmerge, mmark );
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

bool cspray::solver_cg_bubble(charray A, farray x, farray b, int n)
{
	//todo: build precondition
	// 	checkGridFarray( b );
	// 	printf( "b\n" );

	float a, zp, error2, eps = ((float)(1e-2))*n, alpha, beta;
	// z = applyA(x)
	computeAx_bubble << <gsblocknum, threadnum >> >(z, A, x, n);
	// r = b-Ax = b-1.0*z
	pcg_op_bubble << <gsblocknum, threadnum >> >(A, r, b, z, -1.0, n);

	error2 = product(r, r, n);
	if (error2<eps)
		return true;
	//error2 = product( r, r, n );	//error2 = r * r

	//todo: applyPreconditioner
	//buildprecondition_pcg<<<gsblocknum,threadnum>>>( pre, A, z,r,n);
	hipMemcpy(z.data, r.data, sizeof(float)*n, hipMemcpyDeviceToDevice);	//z=r;

	hipMemcpy(p.data, z.data, sizeof(float)*n, hipMemcpyDeviceToDevice);	//p=z;

	a = product(z, r, n);	//a = z*r;

	int k = 0;
	for (; k<n; k++)
	{
		//for debug.
		if (k >= 800 && k % 100 == 0)
		{
			if (!verifyfloat(a) || !verifyfloat(alpha) || !verifyfloat(error2))
				printf("there is some illegal float number in PCG solver!!!\n");
			printf("pcg iteration times: %d\n", k);
			mpause = true;
			break;
		}

		//z=A*p
		computeAx_bubble << <gsblocknum, threadnum >> >(z, A, p, n);

		zp = product(z, p, n);	//zp = z*p
		if (zp == 0)
			return true;

		alpha = a / zp;		//alpha = a/(z . p) = z*r/(z*p)

		// x = x + alpha*p
		pcg_op_bubble << <gsblocknum, threadnum >> >(A, x, x, p, alpha, n);
		// r = r - alpha*z;
		pcg_op_bubble << <gsblocknum, threadnum >> >(A, r, r, z, -alpha, n);

		//error2 = r * r
		error2 = product(r, r, n);
		if (error2<eps)
			break;

		//todo: applyPreconditioner
		hipMemcpy(z.data, r.data, sizeof(float)*n, hipMemcpyDeviceToDevice);	//z=r;
		//buildprecondition_pcg<<<gsblocknum,threadnum>>>( pre, A, z,r,n);

		//a2 = z*r;
		float a2 = product(z, r, n);

		beta = a2 / a;                     // beta = a2 / a
		// p = z + beta*p
		pcg_op_bubble << <gsblocknum, threadnum >> >(A, p, z, p, beta, n);

		a = a2;
	}
	// 	if( mtime )
	// 		printf("CG interation: %d\n", k );

	return true;
}

void cspray::mapvelg2p_bubble()
{
	if (velmode == FLIP)	//in CIP mode, vel of partciles will be updated in advect part.
	{		//ע�⣬��Ҫ��һ���ٶȵĸı䷽ʽ����delta_v������v
		computeDeltaU << <gvblocknum, threadnum >> >(waterux, wateruy, wateruz, waterux_old, wateruy_old, wateruz_old);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");

		computeDeltaU << <gvblocknum, threadnum >> >(airux, airuy, airuz, airux_old, airuy_old, airuz_old);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");

		mapvelg2p_flip_bubble << <pblocknum, threadnum >> >(mParPos, mParVel, parflag, parNumNow, waterux_old, wateruy_old, wateruz_old, airux_old, airuy_old, airuz_old);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");
	}
}

inline float trilinear(farray u, float x, float y, float z, int w, int h, int d)
{
	x = fmax(0.0f, fmin(x, w));
	y = fmax(0.0f, fmin(y, h));
	z = fmax(0.0f, fmin(z, d));
	int i = fmin(x, w - 2);
	int j = fmin(y, h - 2);
	int k = fmin(z, d - 2);

	return (k + 1 - z)*((j + 1 - y)*((i + 1 - x)*u(i, j, k) + (x - i)*u(i + 1, j, k)) + (y - j)*((i + 1 - x)*u(i, j + 1, k) + (x - i)*u(i + 1, j + 1, k))) +
		(z - k)*((j + 1 - y)*((i + 1 - x)*u(i, j, k + 1) + (x - i)*u(i + 1, j, k + 1)) + (y - j)*((i + 1 - x)*u(i, j + 1, k + 1) + (x - i)*u(i + 1, j + 1, k + 1)));
}

float3 cspray::getParticleVelFromGrid(float3 pos, farray ux, farray uy, farray uz)
{
	float3 vel;
	float x = pos.x, y = pos.y, z = pos.z;
	x /= hparam.cellsize.x;
	y /= hparam.cellsize.y;
	z /= hparam.cellsize.z;

	//ע�⣺ux,uy,uz�Ĵ洢��ʽ�Ƚ�����(staggered grid)����ά���Բ�ֵҲҪ�Ƚ�С��
	vel.x = trilinear(ux, x, y - 0.5f, z - 0.5f, NX + 1, NY, NZ);
	vel.y = trilinear(uy, x - 0.5f, y, z - 0.5f, NX, NY + 1, NZ);
	vel.z = trilinear(uz, x - 0.5f, y - 0.5f, z, NX, NY, NZ + 1);
	return vel;
}

void cspray::advect_bubble_CPU()
{

	hipMemcpy(hpos, mParPos, parNumNow*sizeof(float3), hipMemcpyDeviceToHost);
	hipMemcpy(hvel, mParVel, parNumNow*sizeof(float3), hipMemcpyDeviceToHost);
	hipMemcpy(hwaterux.data, waterux.data, hparam.gvnum.x*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hwateruy.data, wateruy.data, hparam.gvnum.y*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hwateruz.data, wateruz.data, hparam.gvnum.z*sizeof(float), hipMemcpyDeviceToHost);

	CTimer time;
	time.startTimer();

	for (int idx = 0; idx<parNumNow; idx++)
	{
		float3 ipos = hpos[idx], ivel = hvel[idx];

		float3 gvel = getParticleVelFromGrid(ipos, hwaterux, hwateruy, hwateruz);

		float3 midpoint = ipos + gvel * hparam.dt * 0.5f;
		float3 gvelmidpoint = getParticleVelFromGrid(midpoint, hwaterux, hwateruy, hwateruz);
		ipos += gvelmidpoint * hparam.dt;

		hvel[idx] = ivel;
		hpos[idx] = ipos;
	}

	printTime(m_bCPURun, "advect_bubble_CPU", time);
}

void cspray::advect_bubble()
{

	if (m_bCPURun)
		advect_bubble_CPU();

	CTimer time;
	time.startTimer();
	advectparticle_RK2_bubble << <pblocknum, threadnum >> >(mParPos, mParVel, parNumNow,
		waterux, wateruy, wateruz, airux, airuy, airuz, hparam.dt, parflag, velmode);

	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
	printTime(m_bCPURun, "advect_bubble", time);
}

bool cspray::solver_cg_heat(charray A, farray x, farray b, int n)
{
	float a, zp, error2, eps = ((float)(1e-2))*n, alpha, beta;
	// z = applyA(x)
	computeAx_heat << <gsblocknum, threadnum >> >(z, A, x, n, HeatAlphaArray, fixedHeat, mscene);
	// r = b-Ax = b-1.0*z
	pcg_op_heat << <gsblocknum, threadnum >> >(A, r, b, z, -1.0, n);

	error2 = product(r, r, n);
	if (error2<eps)
		return true;

	//todo: applyPreconditioner
	//buildprecondition_pcg<<<gsblocknum,threadnum>>>( pre, A, z,r,n);
	hipMemcpy(z.data, r.data, sizeof(float)*n, hipMemcpyDeviceToDevice);	//z=r;

	hipMemcpy(p.data, z.data, sizeof(float)*n, hipMemcpyDeviceToDevice);	//p=z;

	a = product(z, r, n);	//a = z*r;

	int k = 0;
	for (; k<n; k++)
	{
		//for debug.
		if (k >= 3800)
		{
			if (!verifyfloat(a) || !verifyfloat(alpha) || !verifyfloat(error2))
				printf("there is some illegal float number in PCG solver!!!\n");
			printf("pcg iteration times: %d\n", k);
			mpause = true;
			break;
		}

		//z=A*p
		computeAx_heat << <gsblocknum, threadnum >> >(z, A, p, n, HeatAlphaArray, fixedHeat, mscene);

		zp = product(z, p, n);	//zp = z*p
		if (zp == 0)
			return true;

		alpha = a / zp;		//alpha = a/(z . p) = z*r/(z*p)

		// x = x + alpha*p
		pcg_op_heat << <gsblocknum, threadnum >> >(A, x, x, p, alpha, n);
		// r = r - alpha*z;
		pcg_op_heat << <gsblocknum, threadnum >> >(A, r, r, z, -alpha, n);

		//error2 = r * r
		error2 = product(r, r, n);
		if (error2<eps)
		{
			//	printf( "exit: k=%d,error2=%f\n", k,error2 );
			break;
		}

		//todo: applyPreconditioner
		hipMemcpy(z.data, r.data, sizeof(float)*n, hipMemcpyDeviceToDevice);	//z=r;
		//buildprecondition_pcg<<<gsblocknum,threadnum>>>( pre, A, z,r,n);

		//a2 = z*r;
		float a2 = product(z, r, n);

		beta = a2 / a;                     // beta = a2 / a
		// p = z + beta*p
		pcg_op_heat << <gsblocknum, threadnum >> >(A, p, z, p, beta, n);

		a = a2;
	}
	// 	if( mtime )
	// 		printf("CG interation: %d\n", k );

	return true;
}

void cspray::compTpChange_CPU()
{
	//	Tp, Tp_save, mmark 
	static farray htp, htpsave;
	static charray hmark;
	static bool first = true;

	if (first)
	{
		htp.setdim(NX, NY, NZ);
		htpsave.setdim(NX, NY, NZ);
		hmark.setdim(NX, NY, NZ);
		htp.data = new float[NX*NY*NZ];
		htpsave.data = new float[NX*NY*NZ];
		hmark.data = new char[NX*NY*NZ];
	}
	hipMemcpy(htp.data, Tp.data, sizeof(float)*hparam.gnum, hipMemcpyDeviceToHost);
	hipMemcpy(htpsave.data, Tp_save.data, sizeof(float)*hparam.gnum, hipMemcpyDeviceToHost);
	hipMemcpy(hmark.data, mmark.data, sizeof(char)*hparam.gnum, hipMemcpyDeviceToHost);


	CTimer time;
	time.startTimer();
	for (int i = 0; i<hparam.gnum; i++)
	{
		if (hmark[i] != TYPEBOUNDARY)
			htpsave[i] = htp[i] - htpsave[i];
		else
			htpsave[i] = 0;
	}
	printTime(true, "compTpChange_CPU", time);

	first = false;
}

void cspray::updateTemperature()
{
	//1. map heat from particle to grid
	mapHeatp2g_hash << <gsblocknum, threadnum >> >(mParPos, parTemperature, parNumNow, Tp, gridstart, gridend, defaulttemperature);
	hipDeviceSynchronize();

	//0. update the fixed heat with time.
	// 	updateFixedHeat<<<gsblocknum, threadnum>>> ( fixedHeat, mframe );
	// 	hipDeviceSynchronize();

	if (m_bAddHeatBottom)
	{
		addHeatAtBottom << <gsblocknum, threadnum >> >(Tp, mframe, heatIncreaseBottom);
		hipDeviceSynchronize();
	}
	//	setBoundaryHeat<<<gsblocknum, threadnum>>>( Tp );

	hipMemcpy(Tp_save.data, Tp.data, sizeof(float)*hparam.gnum, hipMemcpyDeviceToDevice);
	// 	printf("before cg heat:\n");
	// 	checkGridFarray( Tp );

	//2. set the right side of heat equation
	compb_heat << <gsblocknum, threadnum >> >(Tp_old, Tp, fixedHeat, mmark, HeatAlphaArray);
	hipDeviceSynchronize();

	// 	printf("compb_heat:\n");
	// 	checkGridFarray( Tp_old );

	//3. solve heat
	hipMemset(Tp.data, 0, sizeof(float)*hparam.gnum);	//todo: Ҫ��Ҫ��һ������
	solver_cg_heat(mmark, Tp, Tp_old, hparam.gnum);
	hipDeviceSynchronize();

	//4. set boundary.
	if (m_bExtendHeatToBoundary)
		setBoundaryHeat << <gsblocknum, threadnum >> >(Tp);

	{
		if (m_bCPURun)
			compTpChange_CPU();

		CTimer time;
		time.startTimer();
		compTpChange << <gsblocknum, threadnum >> >(Tp, Tp_save, mmark);
		hipDeviceSynchronize();
		printTime(m_bCPURun, "compTpChange", time);
	}

	//5. map heat from grid to particle
	if (mscene == SCENE_MELTANDBOIL || mscene == SCENE_MELTANDBOIL_HIGHRES)
		mapHeatg2p_MeltAndBoil << <pblocknum, threadnum >> >(mParPos, parflag, parTemperature, parNumNow, Tp_save, Tp, defaultSolidT, alphaTempTrans);
	else
		mapHeatg2p << <pblocknum, threadnum >> >(mParPos, parflag, parTemperature, parNumNow, Tp_save, Tp, defaultSolidT, alphaTempTrans);

	hipDeviceSynchronize();

	updateLatentHeat();
	getLastCudaError("Kernel execution failed");
}

void cspray::initheat_grid()
{
	initheat_grid_k << <gsblocknum, threadnum >> >(Tp, mmark);
	hipDeviceSynchronize();
	markBoundaryCell << <gsblocknum, threadnum >> >(mmark);
	hipDeviceSynchronize();
}


//ר��Ϊ�˱���heat transfer�����һ������
void cspray::heatsim()
{
	if (!mpause)
	{
		CTimer time;
		time.startTimer();
		static CTimer timetotal;
		printTime(m_btimer, "TOTAL TIME!!", timetotal);
		printf("\n------------Frame %d:-------------\n", mframe);

		hipMemcpy(Tp_save.data, Tp.data, sizeof(float)*hparam.gnum, hipMemcpyDeviceToDevice);

		//2. set the right side of heat equation
		compb_heat << <gsblocknum, threadnum >> >(Tp_old, Tp, fixedHeat, mmark, HeatAlphaArray);
		hipDeviceSynchronize();

		//3. solve heat
		hipMemset(Tp.data, 0, sizeof(float)*hparam.gnum);	//todo: Ҫ��Ҫ��һ������
		solver_cg_heat(mmark, Tp, Tp_old, hparam.gnum);
		hipDeviceSynchronize();

		//4. set boundary.
		if (m_bExtendHeatToBoundary)
			setBoundaryHeat << <gsblocknum, threadnum >> >(Tp);

		mframe++;
	}

}

void cspray::initTemperature()
{
	initHeatParticle << <pblocknum, threadnum >> >(parTemperature, parLHeat, defaultSolidT, defaultLiquidT, LiquidHeatTh, parflag, parNumNow);
	hipDeviceSynchronize();
}

void cspray::initSolubility()
{
	initsolubility_k << <pblocknum, threadnum >> >(parsolubility, pargascontain, parTemperature, parflag, parNumNow, 1.0f, Temperature0, initdissolvegasrate, initgasrate);
	hipDeviceSynchronize();
}

void cspray::computesurfacetension()
{
	compsurfacetension_k << <gsblocknum, threadnum >> >(surfacetension, mmark, phigrax, phigray, phigraz, surfacetensionsigma);
	hipDeviceSynchronize();

	enforcesurfacetension_p << <pblocknum, threadnum >> >(mParPos, mParVel, parflag, parNumNow, lsmerge, surfacetension, phigrax, phigray, phigraz, mmark, mscene);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::sweepLSAndMardGrid()
{
	//fluid
	//��sweep��ֵ����sweep��ֵ
	markLS_bigpositive << <gsblocknum, threadnum >> >(lsfluid, mmark);
	hipDeviceSynchronize();
	for (int it = 0; it<5; it++)
	{
		sweepphibytype << <gsblocknum, threadnum >> >(lsfluid, mmark, TYPEFLUID);
		hipDeviceSynchronize();
	}
	setLSback_bigpositive << <gsblocknum, threadnum >> >(lsfluid);
	hipDeviceSynchronize();

	preparels << <gsblocknum, threadnum >> >(lsfluid, mmark);
	hipDeviceSynchronize();
	for (int it = 0; it<5; it++)
	{
		sweepphibytype << <gsblocknum, threadnum >> >(lsfluid, mmark, TYPEFLUID);
		hipDeviceSynchronize();
	}
	setLSback << <gsblocknum, threadnum >> >(lsfluid);
	hipDeviceSynchronize();

	//air
	//��sweep��ֵ����sweep��ֵ
	markLS_bigpositive << <gsblocknum, threadnum >> >(lsair, mmark);
	hipDeviceSynchronize();
	for (int it = 0; it<5; it++)
	{
		sweepphibytype << <gsblocknum, threadnum >> >(lsair, mmark, TYPEFLUID);
		hipDeviceSynchronize();
	}
	setLSback_bigpositive << <gsblocknum, threadnum >> >(lsair);
	hipDeviceSynchronize();

	preparels << <gsblocknum, threadnum >> >(lsair, mmark);
	hipDeviceSynchronize();
	for (int it = 0; it<5; it++)
	{
		sweepphibytype << <gsblocknum, threadnum >> >(lsair, mmark, TYPEFLUID);
		hipDeviceSynchronize();
	}
	setLSback << <gsblocknum, threadnum >> >(lsair);
	hipDeviceSynchronize();

	//merge the level set.
	mergeLSAndMarkGrid << <gsblocknum, threadnum >> >(lsmerge, mmark, lsfluid, lsair);
	hipDeviceSynchronize();

	//������볡���ݶ�
	computePhigra << <gsblocknum, threadnum >> >(phigrax, phigray, phigraz, lsmerge);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");
}

void cspray::correctpos_bubble()
{
	static bool first = true;
	static float *dphi;
	//	char *hflag;
	if (first)
	{
		hipMalloc((void**)&dphi, sizeof(float)*parNumMax);
		// 		hphi=new float[parNumMax];
		// 		hflag = new char[parNumMax];
		first = false;
	}
	hipMemset(dphi, 0, parNumNow*sizeof(float));


	if (mscene == SCENE_MELTANDBOIL || mscene == SCENE_MELTANDBOIL_HIGHRES || mscene == SCENE_INTERACTION)
	{
		computePhigra << <gsblocknum, threadnum >> >(phigrax_air, phigray_air, phigraz_air, lsair);
		correctbubblepos_air << <pblocknum, threadnum >> >(lsmerge, phigrax, phigray, phigraz, lsair, phigrax_air, phigray_air, phigraz_air,
			mParPos, parflag, parNumNow, dphi);
	}
	else
		correctbubblepos << <pblocknum, threadnum >> >(lsmerge, phigrax, phigray, phigraz, mParPos, parflag, parNumNow, dphi);


	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed");

	//	first=false;
	// 	hipMemcpy( hphi, dphi, parNumNow*sizeof(float), hipMemcpyDeviceToHost );
	// 	hipMemcpy( hflag, parflag, parNumNow*sizeof(char), hipMemcpyDeviceToHost );
	// 	float aver=0, maxphi=-100000, minphi=1000000;
	// 	for( int i=0; i<parNumNow; i++ )
	// 	{
	// 		float t=hphi[i]/hparam.cellsize.x;
	// 		if( hflag[i]==TYPEAIR )
	// 			printf( "phi=%f\n", t ), maxphi=max(maxphi,t), minphi=min(minphi,t);
	// 	}
	// 	printf( "%f, %f\n", maxphi, minphi );
	// 
}

void cspray::initscene_fluidsphere()
{
	if (parNumNow <= 0)
		return;

	float3* hparpos = new float3[parNumNow];
	float3* hparvel = new float3[parNumNow];
	float* hparmass = new float[parNumNow];
	char* hparflag = new char[parNumNow];

	float3 bubblepos1 = make_float3(NX*0.5f, NY*0.5f, NZ*0.4f) * hparam.cellsize.x;		//bottom
	float3 bubblepos2 = make_float3(NX*0.5f, NY*0.5f, NZ*0.6f) * hparam.cellsize.x;		//bottom
	float3 bubblepos3 = make_float3(NX*0.5f, NY*0.5f, NZ*0.75f) * hparam.cellsize.x;		//bottom
	float3 bubblepos4 = make_float3(NX*0.3f, NY*0.5f, NZ*0.15f) * hparam.cellsize.x;		//bottom
	float3 bubblepos5 = make_float3(NX*0.7f, NY*0.5f, NZ*0.15f) * hparam.cellsize.x;		//bottom
	float bubbleradius = NX*0.5f*hparam.cellsize.x * 0.3f;		//small bubble
	float3 temppos;

	int i = 0;
	for (float z = hparam.cellsize.x + hparam.samplespace; z< (NZ - 2)*hparam.cellsize.x && i<initfluidparticle; z += hparam.samplespace)
	for (float y = hparam.cellsize.x + hparam.samplespace; y<hparam.cellsize.x*(NY - 1) - 0.5f*hparam.samplespace && i<initfluidparticle; y += hparam.samplespace)
	for (float x = hparam.cellsize.x + hparam.samplespace; x<hparam.cellsize.x*(NX - 1) - 0.5f*hparam.samplespace && i<initfluidparticle; x += hparam.samplespace)
	{
		temppos = make_float3(0.4f*(x - bubblepos1.x), y - bubblepos1.y, z - bubblepos1.z);
		if (length(temppos)<bubbleradius)
		{
			hparflag[i] = TYPEFLUID;
			hparpos[i] = make_float3(x, y, z);
			hparvel[i] = make_float3(0.0f);
			hparmass[i] = hparam.m0;
			i++;
		}
		temppos = make_float3(0.6f*(x - bubblepos2.x), y - bubblepos2.y, z - bubblepos2.z);
		if (length(temppos)<bubbleradius)
		{
			hparflag[i] = TYPEFLUID;
			hparpos[i] = make_float3(x, y, z);
			hparvel[i] = make_float3(0.0f);
			hparmass[i] = hparam.m0;
			i++;
		}
		temppos = make_float3(1.0f*(x - bubblepos3.x), y - bubblepos3.y, z - bubblepos3.z);
		if (length(temppos)<bubbleradius)
		{
			hparflag[i] = TYPEFLUID;
			hparpos[i] = make_float3(x, y, z);
			hparvel[i] = make_float3(0.0f);
			hparmass[i] = hparam.m0;
			i++;
		}
		temppos = make_float3((x - bubblepos4.x), y - bubblepos4.y, z - bubblepos4.z)*0.75f;
		if (length(temppos)<bubbleradius)
		{
			hparflag[i] = TYPEFLUID;
			hparpos[i] = make_float3(x, y, z);
			hparvel[i] = make_float3(0.0f);
			hparmass[i] = hparam.m0;
			i++;
		}
		temppos = make_float3((x - bubblepos5.x), y - bubblepos5.y, z - bubblepos5.z)*0.75f;
		if (length(temppos)<bubbleradius)
		{
			hparflag[i] = TYPEFLUID;
			hparpos[i] = make_float3(x, y, z);
			hparvel[i] = make_float3(0.0f);
			hparmass[i] = hparam.m0;
			i++;
		}
	}

	parNumNow = i;
	//debug:
	//hparpos[parNumNow-1] = make_float3(1-1.2f/64,1.2f/64, 1-1.2f/64 );

	hipMemcpy(mParPos, hparpos, sizeof(float3)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(mParVel, hparvel, sizeof(float3)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(parmass, hparmass, sizeof(float)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(parflag, hparflag, sizeof(char)*parNumNow, hipMemcpyHostToDevice);

	delete[] hparpos;
	delete[] hparvel;
	delete[] hparmass;
	delete[] hparflag;
}

void cspray::deleteAirFluidParticle()
{
	//1. sweep vacuum, mark the air cell as vacuum if it adjoins a vacuum cell.
	if (mscene == SCENE_INTERACTION)
	{

		for (int t = 0; t<1 && mframe % 2 == 0; t++)
			sweepVacuum << <gsblocknum, threadnum >> >(mmark);
	}
	else
	{
		for (int t = 0; t<20; t++)
			sweepVacuum << <gsblocknum, threadnum >> >(mmark);
	}

	//2. mark the deleted particle
	uint *cnt;
	hipMalloc((void**)&cnt, sizeof(uint));
	hipMemset(cnt, 0, sizeof(uint));
	markDeleteAirParticle << <pblocknum, threadnum >> >(mParPos, parflag, parmass, preservemark, parNumNow, mmark, lsmerge, lsair, cnt);
	uint *hcnt = new uint[1];
	hipMemcpy(hcnt, cnt, sizeof(uint), hipMemcpyDeviceToHost);
	static uint totalCnt = 0;
	totalCnt += hcnt[0];
	printf("totalCnt for deleting fluid particle=%u\n", totalCnt);

	//3. delete particle.
	//3. scan���õ���Ҫ���������ӵ��������
	int activeParticleNum = 0;
	ThrustScanWrapper(preservemarkscan, preservemark, parNumNow);
	{
		uint lastElement, lastScanElement;
		checkCudaErrors(hipMemcpy((void *)&lastElement,
			(void *)(preservemark + parNumNow - 1),
			sizeof(uint), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy((void *)&lastScanElement,
			(void *)(preservemarkscan + parNumNow - 1),
			sizeof(uint), hipMemcpyDeviceToHost));
		activeParticleNum = lastElement + lastScanElement;
	}

	if (activeParticleNum == parNumNow)	//���û����Ҫɾ��������
		return;

	//4. copy�������ӵ��������ٶȡ���ǡ�λ�ÿ���������
	swapParticlePointers();
	deleteparticles << <pblocknum, threadnum >> >(preservemark, preservemarkscan, parNumNow,
		mParPos, tmpParPos, mParVel, tmpParVelFLIP, parmass, tmpparmass, parflag, tmpparflag, parTemperature, tmpparTemperature,
		parLHeat, tmpparHeat, parsolubility, tempsolubility, pargascontain, tempgascontain);

	//5. �޸����ӵĸ���������pblocknum
	parNumNow = activeParticleNum;
	pblocknum = max(1, (int)ceil(((float)parNumNow) / threadnum));
	printf("After deleting: particle number:%d\n", parNumNow);
}

void cspray::outputPovRaywater(int frame, float3* dpos, float3 *dnormal, int pnum, uint *dindices, int indicesnum, const char* objectname)
{
	//filename
	static char filename[100];
	sprintf(filename, "%swaterdata\\%s%05d.pov", outputdir, objectname, frame);
	FILE *fp = fopen(filename, "w");
	if (fp == NULL)
	{
		printf("cannot open pov file for output!!\n");
		mpause = true;
		return;
	}

	//���û��MC�����Σ���ֻ����ͷ�ļ������
	if (pnum == 0)
	{
		fclose(fp);
		return;
	}

	fprintf(fp, "#declare watermesh=mesh2{\n");
	fprintf(fp, "vertex_vectors{\n");
	fprintf(fp, "%d,\n", pnum);

	// vertex positions
	static float3* hpos = new float3[maxVerts];
	hipMemcpy(hpos, dpos, pnum*sizeof(float3), hipMemcpyDeviceToHost);
	for (int i = 0; i<pnum; i++)
	{
		if (!verifyfloat3(hpos[i]))
			hpos[i] = make_float3(0.0f);
		fprintf(fp, "< %f, %f, %f >,\n", hpos[i].x, hpos[i].y, hpos[i].z);
	}
	fprintf(fp, "}\n ");

	fprintf(fp, "normal_vectors{\n");
	fprintf(fp, "%d,\n", pnum);
	//vertex normals
	hipMemcpy(hpos, dnormal, pnum*sizeof(float3), hipMemcpyDeviceToHost);
	for (int i = 0; i<pnum; i++)
	{
		if (!verifyfloat3(hpos[i]))
			hpos[i] = make_float3(0.0f);
		fprintf(fp, "<%f, %f, %f>,\n", hpos[i].x, hpos[i].y, hpos[i].z);
	}
	fprintf(fp, "}\n ");

	fprintf(fp, "face_indices{\n");
	fprintf(fp, "%d,\n", indicesnum / 3);
	//face indices.
	static uint *hindices = new uint[MCedgeNum];
	hipMemcpy(hindices, dindices, indicesnum*sizeof(uint), hipMemcpyDeviceToHost);
	for (int i = 0; i<indicesnum; i += 3)
		fprintf(fp, "<%u, %u, %u>,\n", hindices[i], hindices[i + 1], hindices[i + 2]);	//povray indices from 0 !!!
	fprintf(fp, "}\n ");

	fprintf(fp, "inside_vector <0,0,1> }\n ");
	fprintf(fp, "object{ watermesh material{%s_material} }\n ", objectname);

	fclose(fp);
}
void cspray::outputColoredParticle(int frame, float3* dpos, float *dtemperature, int pnum)
{
	static float3 *hpos = new float3[parNumMax];
	static float *htemperature = new float[parNumMax];

	hipMemcpy(hpos, dpos, pnum*sizeof(float3), hipMemcpyDeviceToHost);
	hipMemcpy(htemperature, dtemperature, pnum*sizeof(float), hipMemcpyDeviceToHost);

	//filename
	static char filename[100];
	sprintf(filename, "%swaterdata\\allparticles%05d.pov", outputdir, frame);
	FILE *fp = fopen(filename, "w");
	if (fp == NULL)
	{
		printf("cannot open pov file for output!!\n");
		return;
	}

	float iradius = 0.004f;
	float3 color;
	for (int i = 0; i<pnum; i++)
	{
		color = mapColorBlue2Red_h((htemperature[i] - temperatureMin_render) / (temperatureMax_render - temperatureMin_render)*6.0f);
		fprintf(fp, "sphere{ <%f,%f,%f> %f texture{ finish{dropletFinish} pigment{ rgb<%f,%f,%f>}} }\n", hpos[i].x, hpos[i].y, hpos[i].z, iradius, color.x, color.y, color.z);
	}

	fclose(fp);
}

void cspray::outputSoloBubblePovRay(int frame, float3 *dpos, float *dmass, char *dflag, int pnum)
{
	static float3 *hpos = new float3[parNumMax];
	//	static float *hmass = new float[parNumMax];
	static char *hflag = new char[parNumMax];

	hipMemcpy(hpos, dpos, pnum*sizeof(float3), hipMemcpyDeviceToHost);
	//	hipMemcpy( hmass, dmass, pnum*sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy(hflag, dflag, pnum*sizeof(char), hipMemcpyDeviceToHost);

	//filename
	static char filename[100];
	sprintf(filename, "%swaterdata\\solobubble%05d.pov", outputdir, frame);
	FILE *fp = fopen(filename, "w");
	if (fp == NULL)
	{
		printf("cannot open pov file for output!!\n");
		return;
	}

	float iradius = 0.005f;
	for (int i = 0; i<pnum; i++)
	{
		if (hflag[i] == TYPEAIRSOLO)
			fprintf(fp, "sphere{ <%f,%f,%f> %f texture{bblTexture} }\n", hpos[i].x, hpos[i].y, hpos[i].z, iradius);
	}

	fclose(fp);
}

void cspray::outputAirParticlePovRay(int frame, float3 *dpos, float *dmass, char *dflag, int pnum)
{
	static float3 *hpos = new float3[parNumMax];
	//	static float *hmass = new float[parNumMax];
	static char *hflag = new char[parNumMax];

	hipMemcpy(hpos, dpos, pnum*sizeof(float3), hipMemcpyDeviceToHost);
	//	hipMemcpy( hmass, dmass, pnum*sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy(hflag, dflag, pnum*sizeof(char), hipMemcpyDeviceToHost);

	//filename
	static char filename[100];
	sprintf(filename, "%swaterdata\\solobubble%05d.pov", outputdir, frame);
	FILE *fp = fopen(filename, "w");
	if (fp == NULL)
	{
		printf("cannot open pov file for output!!\n");
		return;
	}

	float iradius = 0.005f;
	for (int i = 0; i<pnum; i++)
	{
		if (hflag[i] == TYPEAIR)
			fprintf(fp, "sphere{ <%f,%f,%f> %f texture{bblTexture} }\n", hpos[i].x, hpos[i].y, hpos[i].z, iradius);
	}

	fclose(fp);
}

void cspray::outputEmptyBubblePovRay(int frame)
{
	//filename
	static char filename[100];
	sprintf(filename, "%swaterdata\\emptybubble%05d.pov", outputdir, frame);
	FILE *fp = fopen(filename, "w");
	if (fp == NULL)
	{
		printf("cannot open pov file for output!!\n");
		return;
	}

	static float3 *hEmptyPos = new float3[pEmptyNum];
	static float3 *hEmptyDir = new float3[pEmptyNum];
	static float *hEmptyRadius = new float[pEmptyNum];
	hipMemcpy(hEmptyPos, pEmptyPos, pEmptyNum*sizeof(float3), hipMemcpyDeviceToHost);
	hipMemcpy(hEmptyDir, pEmptyDir, pEmptyNum*sizeof(float3), hipMemcpyDeviceToHost);
	hipMemcpy(hEmptyRadius, pEmptyRadius, pEmptyNum*sizeof(float), hipMemcpyDeviceToHost);
	printf("emptybubble pos=%f,%f,%f, radius=%f\n", hEmptyPos[0].x, hEmptyPos[0].y, hEmptyPos[0].z, hEmptyRadius[0]);

	//	float iradius=0.005f;
	for (int i = 0; i<pEmptyNum; i++)
	{
		//if( hflag[i]==TYPEAIRSOLO)
		fprintf(fp, "sphere{ <%f,%f,%f> %f texture{bblTexture} }\n", hEmptyPos[i].x, hEmptyPos[i].y, hEmptyPos[i].z, hEmptyRadius[0]);
	}

	fclose(fp);
}


void cspray::initscene_multibubble()
{
	if (parNumNow <= 0)
		return;

	float3* hparpos = new float3[parNumNow];
	float3* hparvel = new float3[parNumNow];
	float* hparmass = new float[parNumNow];
	char* hparflag = new char[parNumNow];

	float bubbleradius = hparam.cellsize.x*3.0f;		//small bubble
	const int bubblecnt = 3;
	float3 bubblepos[bubblecnt];
	int k = 0;
	for (int i = 1; i <= 3; ++i)
		bubblepos[k++] = make_float3(i / 4.0f*NX, 0.5f*NY, 5.0f)*hparam.cellsize.x;
	float3 temppos;

	int i = 0;
	for (float z = hparam.cellsize.x + hparam.samplespace; z< (NZ - 2)*hparam.cellsize.x && i<initfluidparticle; z += hparam.samplespace)
	{
		for (float y = hparam.cellsize.x + hparam.samplespace; y<hparam.cellsize.x*(NY - 1) - 0.5f*hparam.samplespace && i<initfluidparticle; y += hparam.samplespace)
		for (float x = hparam.cellsize.x + hparam.samplespace; x<hparam.cellsize.x*(NX - 1) - 0.5f*hparam.samplespace && i<initfluidparticle; x += hparam.samplespace)
		{
			bool flag = false;
			for (int cnt = 0; cnt<bubblecnt; ++cnt)
			{
				temppos = make_float3(x, y, z);
				if (length(temppos - bubblepos[cnt])<bubbleradius)
					flag = true;
			}
			if (flag)
				hparflag[i] = TYPEAIRSOLO;
			else
				hparflag[i] = TYPEFLUID;
			hparpos[i] = make_float3(x, y, z);
			hparvel[i] = make_float3(0.0f);
			hparmass[i] = hparam.m0;
			i++;
		}
	}

	parNumNow = i;
	//debug:
	//hparpos[parNumNow-1] = make_float3(1-1.2f/64,1.2f/64, 1-1.2f/64 );

	hipMemcpy(mParPos, hparpos, sizeof(float3)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(mParVel, hparvel, sizeof(float3)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(parmass, hparmass, sizeof(float)*parNumNow, hipMemcpyHostToDevice);
	hipMemcpy(parflag, hparflag, sizeof(char)*parNumNow, hipMemcpyHostToDevice);

	delete[] hparpos;
	delete[] hparvel;
	delete[] hparmass;
	delete[] hparflag;
}

void cspray::markSoloAirParticle()
{
	verifySoloAirParticle << <pblocknum, threadnum >> >(mParPos, mParVel, parflag, parNumNow, lsmerge, airux, airuy, airuz, gridstart, gridend, mscene);
	hipDeviceSynchronize();
}

void cspray::updateSoloAirParticle()
{
	calcDensPress_Air << <pblocknum, threadnum >> >(mParPos, pardens, parpress, parflag, parNumNow, gridstart, gridend);
	hipDeviceSynchronize();

	//debug
	if (0)
	{
		static float *hdens = new float[parNumMax];
		static char *hflag = new char[parNumMax];
		hipMemcpy(hdens, pardens, parNumNow*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(hflag, parflag, parNumNow*sizeof(char), hipMemcpyDeviceToHost);

		float densaver = 0, densmax = -1, densmin = 10000000;
		int cnt = 0;
		for (int i = 0; i<parNumNow; i++)
		{
			if (hflag[i] == TYPEAIRSOLO) {
				densaver += 1 / hdens[i];
				densmax = max(densmax, 1 / hdens[i]);
				densmin = min(densmin, 1 / hdens[i]);
				cnt++;
				printf("%f, ", 1 / hdens[i]);
			}
		}
		densaver /= cnt;
		printf("densaver, max, min=%f,%f,%f\n", densaver, densmax, densmin);
	}

	enforceForceSoloAirP << <pblocknum, threadnum >> >(mParPos, mParVel, pardens, parpress, parflag, parNumNow, gridstart, gridend, viscosiySPH, maxVelForBubble);
	hipDeviceSynchronize();
}

void cspray::GenerateGasParticle()
{
	//1. �����¶ȸı�ÿ�����ӵ��ܽ��
	updatesolubility << <pblocknum, threadnum >> >(parsolubility, parTemperature, parflag, parNumNow, 1.0f, Temperature0, initdissolvegasrate);
	hipDeviceSynchronize();

	static bool first = true;

	static int *daddparnums;
	int haddparnums = 0;
	//	static bool first=true;
	if (first)
	{
		hipMalloc((void**)&daddparnums, sizeof(int));
		first = false;
	}
	hipMemcpy(daddparnums, &haddparnums, sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	//3. �����ܽ�Ⱥ�latent heat������������
	GenerateGasParticle_k << <gsblocknum, threadnum >> >(parsolubility, pargascontain, mParPos, mParVel, parmass, parflag, parTemperature, parLHeat, parNumNow, gridstart, gridend,
		daddparnums, randfloat, randfloatcnt, mframe, Tp, LiquidHeatTh, dseedcell, seednum, vaporGenRate);

	//update the number of particles.
	hipMemcpy(&haddparnums, daddparnums, sizeof(int), hipMemcpyDeviceToHost);
	parNumNow += haddparnums;
	pblocknum = max(1, (int)ceil(((float)parNumNow) / threadnum));
	//printf("addparnums=%d\n", haddparnums );

	//printf( "After adjustAirParBySolubility, particle number=%d\n", parNumNow );
}

void cspray::solidmotion_fixed()
{
	setVelZeroSolid_k << <pblocknum, threadnum >> >(mParVel, parflag, parNumNow);
	hipDeviceSynchronize();
}

void cspray::solidmotion()			///////////////////////////////
{
	//��û�й���������ֱ�ӷ���
	if (nRealSolpoint <= 0)
		return;

	//	printf("nrealSolPoint=%d\n", nRealSolpoint );

	//0. �����������ǽ�����
	solidCollisionWithBound << <pblocknum, threadnum >> > (mParPos, mParVel, parflag, parNumNow, SolidbounceParam, nRealSolpoint);
	hipDeviceSynchronize();

	//1. prepare
	hipMemcpy(solidParPos, mParPos, sizeof(float3)*parNumNow, hipMemcpyDeviceToDevice);
	hipMemcpy(solidParVelFLIP, mParVel, sizeof(float3)*parNumNow, hipMemcpyDeviceToDevice);

	set_nonsolid_2_zero << <pblocknum, threadnum >> >(parflag, parNumNow, solidParPos, solidParVelFLIP);	//	�ǹ��������ٶȺ�λ����0.�Ϳ���ֱ���ۼ���
	hipDeviceSynchronize();

	float3 T, tmpI;	//T: ���ٶȣ�tmpI��ÿ�����ӵ����ĵľ���
	float3 R, rg;		//R:���ٶȣ�rg: ����

	R = T = rg = make_float3(0, 0, 0);
	float fSolNum = (float)nRealSolpoint;

	//2. ���������ٶ�
	T = accumulate_GPU_f3(solidParVelFLIP);
	//	printf("T=%f,%f,%f\n", T.x, T.y, T.z );
	T = T / fSolNum;

	//interaction������y���仯������۲���������ݵĽ���
	if (mscene == SCENE_INTERACTION)
		T.y = 0.0f;

	//3. ����������
	rg0 = accumulate_GPU_f3(solidParPos) / fSolNum;
	rg = rg0 + hparam.dt*T;					//��������λ��

	//4. �����Ľ��ٶ�
	compute_cI_k << <pblocknum, threadnum >> >(parNumNow, parflag, mParPos, mParVel, c, I, rg0);
	tmpI = accumulate_GPU_f3(I);	//�����ӵ����ĵľ���֮�ͣ��൱�ڽ��ٶȵ�Ȩ��
	R = accumulate_GPU_f3(c);		//���ٶȣ���Ȩ�أ�
	if (tmpI.x <= 0)	tmpI.x = 1;
	//����Ĳ�����Щ��˼��tmpI.x�ĸ�����ȫ�����ӵ����ľ����ƽ��ֵ����R�Ǹ����ӡ����ٶ�(û�г�|r|^2)����ƽ��ֵ��
	//��������������ÿ�����Ӹ��Լ�����ٶ�Ҫ������ȷ�������Բ���һ��ֻ���������³��ĳ�����
	tmpI.x /= fSolNum;
	R = R / tmpI.x / tmpI.x;		//��һ��
	R /= fSolNum;
	//	printf( "tempi.x=%f\n", tmpI.x );

	//printf( "rg=%f,%f,%f, R=%f,%f,%f, T=%f,%f,%f\n", rg.x, rg.y, rg.z, R.x, R.y, R.z, T.x, T.y, T.z );

	//5. ���¸����ӵ��ٶ�
	computeVelSolid_k << <pblocknum, threadnum >> >(mParPos, parflag, mParVel, parNumNow, rg0, R, T);
	hipDeviceSynchronize();

	//6. ���¸����ӵ�λ��
	{
		//debug
		//R=make_float3( 0,0,1 );
		float3 axis = R;		//���ٶȾ�������ת��
		float theta = -length(R)*hparam.dt;		//���ٶȾ�������ת�Ƕȴ�С
		if (abs(theta)>1e-6)		//ע�⣺�᳤Ϊ0ʱ�ǲ���normalize�ģ����г�0�Ĵ���
			axis = normalize(axis);
		else
			axis = make_float3(1, 0, 0), theta = 0;

		matrix3x3 rm;		//��ת��������ת��axis����ת�Ƕ�theta��������ʽ�ο���http://zh.wikipedia.org/wiki/%E6%97%8B%E8%BD%AC%E7%9F%A9%E9%98%B5
		float x = axis.x, y = axis.y, z = axis.z;
		//printf("axis=%f,%f,%f, theta=%f\n", x,y,z,theta);
		float c = cos(theta), s = sin(theta);
		rm.x00 = c + (1 - c)*x*x, rm.x01 = (1 - c)*x*y - s*z, rm.x02 = (1 - c)*x*z + s*y;
		rm.x10 = (1 - c)*y*x + s*z, rm.x11 = c + (1 - c)*y*y, rm.x12 = (1 - c)*y*z - s*x;
		rm.x20 = (1 - c)*z*x - s*y, rm.x21 = (1 - c)*z*y + s*x, rm.x22 = c + (1 - c)*z*z;
		//normalize? not needed.

		computePosSolid_k << <pblocknum, threadnum >> >(mParPos, parflag, parNumNow, rg, rg0, rm);
		hipDeviceSynchronize();

		int blocknum = (int)ceil(((float)solidvertexnum) / threadnum);
		if ((mscene == SCENE_INTERACTION || mscene == SCENE_INTERACTION_HIGHRES || mscene == SCENE_MELTANDBOIL || mscene == SCENE_MELTANDBOIL_HIGHRES) && mframe>0 && !bRunMCSolid)
			computeSolidVertex_k << <blocknum, threadnum >> >(solidvertex, solidvertexnum, rg, rg0, rm);
	}
}

//parNumNow�����ӵ�float3���͵����ݵĹ�Լ���
float3 cspray::accumulate_GPU_f3(float3 *data)
{
	int maxblockNum = max(1, (int)ceil(((float)parNumMax) / threadnum));
	static float3 *ssum = NULL;
	if (!ssum)
		hipMalloc((void**)&ssum, sizeof(float3)*maxblockNum/*GRIDCOUNT(60160,256)*/);
	static float3*hsum = new float3[maxblockNum];
	int sharememsize = threadnum*sizeof(float3);

	accumulate_GPU_k << <pblocknum, threadnum, sharememsize >> >(parNumNow, ssum, data);
	hipDeviceSynchronize();

	hipMemcpy(hsum, ssum, sizeof(float3)*pblocknum, hipMemcpyDeviceToHost);

	float3 res = make_float3(0);
	for (int i = 0; i<pblocknum; i++)
	{
		res += hsum[i];
		//	printf( "debug: hsum = %f,%f,%f,%i\n", hsum[i].x,hsum[i].y,hsum[i].z,i);
	}
	return res;
}

//parNumNow�����ӵ�float3���͵����ݵĹ�Լ���
float3 cspray::accumulate_CPU_f3_test(float3 *data)
{
	float3 res = make_float3(0);
	static float3 *hdata = new float3[parNumMax];
	hipMemcpy(hdata, data, sizeof(float3)*parNumNow, hipMemcpyDeviceToHost);

	for (int i = 0; i<parNumNow; i++)
	{
		res += hdata[i];
		//	printf( "debug: hsum = %f,%f,%f,%i\n", hsum[i].x,hsum[i].y,hsum[i].z,i);
	}
	return res;
}


void cspray::MeltSolid()
{
	static int *dnumchange = NULL;
	if (!dnumchange)
		hipMalloc((void**)&dnumchange, sizeof(int));
	int hnumchange = 0;
	hipMemset(dnumchange, 0, sizeof(int));
	hipDeviceSynchronize();

	MeltingSolidByHeat << <pblocknum, threadnum >> > (parTemperature, parLHeat, parflag, parNumNow, LiquidHeatTh, meltingpoint, dnumchange);
	hipDeviceSynchronize();

	hipMemcpy((void*)&hnumchange, dnumchange, sizeof(int), hipMemcpyDeviceToHost);
	nRealSolpoint -= hnumchange;
	//printf( "the new solid point number = %d\n", nRealSolpoint );
}

void cspray::MeltSolid_CPU()
{
	hipMemcpy(hparflag, parflag, sizeof(char)*parNumNow, hipMemcpyDeviceToHost);
	hipMemcpy(hparLHeat, parLHeat, sizeof(float)*parNumNow, hipMemcpyDeviceToHost);

	CTimer time;
	time.startTimer();
	int hnumchange = 0;

	for (int i = 0; i<parNumNow; i++)
	{
		if (hparflag[i] == TYPESOLID)
		{
			if (hparLHeat[i]>LiquidHeatTh)
			{
				hparflag[i] = TYPEFLUID;
				hparLHeat[i] = LiquidHeatTh;
				hnumchange = 0;
			}
		}
	}

	nRealSolpoint -= hnumchange;
	//printf( "the new solid point number = %d\n", nRealSolpoint );
	printTime(true, "MeltSolid_CPU", time);
}

void cspray::Freezing()
{
	static int *dnumchange = NULL;
	if (!dnumchange)
		hipMalloc((void**)&dnumchange, sizeof(int));
	int hnumchange = 0;
	hipMemset(dnumchange, 0, sizeof(int));
	hipDeviceSynchronize();

	FreezingSolidByHeat << <pblocknum, threadnum >> > (mParPos, parLHeat, parflag, parNumNow, dnumchange, gridstart, gridend);
	hipDeviceSynchronize();

	hipMemcpy((void*)&hnumchange, dnumchange, sizeof(int), hipMemcpyDeviceToHost);
	nRealSolpoint += hnumchange;
	//printf( "Freezing: new solid point number=%d\n", nRealSolpoint );
}

void cspray::initEmptyBubbles()
{
	pEmptyNum = 1;
	float3 *hEmptyPos = new float3[pEmptyNum];
	float3 *hEmptyDir = new float3[pEmptyNum];
	float *hEmptyRadius = new float[pEmptyNum];

	for (int i = 0; i<pEmptyNum; i++)
	{
		hEmptyPos[i] = make_float3(12.f*hparam.cellsize.x, 12.f*hparam.cellsize.x, 1.6f*hparam.cellsize.x);
		hEmptyDir[i] = make_float3(0, 0, 1);
		hEmptyRadius[i] = 0;
	}
	printf("emptybubble pos=%f,%f,%f, radius=%f\n", hEmptyPos[0].x, hEmptyPos[0].y, hEmptyPos[0].z, hEmptyRadius[0]);

	//for empty���ݵ�����
	hipMalloc((void**)&pEmptyPos, sizeof(float3)*pEmptyNum);
	hipMalloc((void**)&pEmptyDir, sizeof(float3)*pEmptyNum);
	hipMalloc((void**)&pEmptyRadius, sizeof(float)*pEmptyNum);

	hipMemcpy(pEmptyPos, hEmptyPos, pEmptyNum*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpy(pEmptyDir, hEmptyDir, pEmptyNum*sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpy(pEmptyRadius, hEmptyRadius, pEmptyNum*sizeof(float), hipMemcpyHostToDevice);

	delete[] hEmptyPos;
	delete[] hEmptyDir;
	delete[] hEmptyRadius;
}


void cspray::initSeedCell()
{
	//���x,y��ֵ��Ȼ�������ӵ�ֵ
	int *hseedcells = new int[seednum];
	int z = 2;
	for (int i = 0; i<seednum; i++)
	{
		bool has = true;	//��¼�������ɵ�cell�Ƿ��Ѿ����ù�
		while (has)
		{
			has = false;
			int x = rand() % (NX - 4) + 2;
			int y = rand() % (NY - 4) + 2;
			int gidx = getidx(x, y, z);
			for (int j = 0; j<i; j++)
			if (hseedcells[j] == gidx) has = true;
			if (!has)
			{
				hseedcells[i] = gidx;
				//	printf("x,y=%d,%d\n", x, y);
			}
		}
	}
	hipMemcpy(dseedcell, hseedcells, sizeof(int)*seednum, hipMemcpyHostToDevice);

	delete[] hseedcells;
}

void cspray::updateSeedCell()
{
	static int idx = 0;

	int cnt = 0, bound = 3;
	while (cnt++ <= 3)
	{
		int z = 2;
		int x = rand() % (NX - 2 * bound) + bound;
		int y = rand() % (NY - 2 * bound) + bound;
		int gidx = getidx(x, y, z);

		hipMemcpy(dseedcell + idx, &gidx, sizeof(int), hipMemcpyHostToDevice);
		idx++;
		idx %= seednum;
	}
}

void cspray::enforceDragForce()
{
	calDragForce << < pblocknum, threadnum >> >(mParPos, mParVel, parflag, parNumNow, waterux, wateruy, wateruz, dragParamSolo, dragParamGrid, mscene);
	hipDeviceSynchronize();
}

void cspray::CollisionSolid()
{
	//1. prepare phi and velocity of solid (only solid, this is for collision)
	initSolidPhi << <gsblocknum, threadnum >> >(phisolid, gridstart, gridend, parflag);
	hipDeviceSynchronize();

	for (int it = 0; it<3; it++)
	{
		sweepphi << <gsblocknum, threadnum >> >(phisolid);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");
	}
	//1.2, velocity
	mapvelp2g_k_fluidSolid << <gvblocknum, threadnum >> >(mParPos, mParVel, parmass, parflag, parNumNow, solidux, soliduy, soliduz, gridstart, gridend);
	hipDeviceSynchronize();

	//todo: sweep u
	//2. modify the velocity of air/airsolo/fluid particle, and update their positions.
	if (mscene == SCENE_FREEZING || mscene == SCENE_MELTINGPOUR)
		CollisionWithSolid_Freezing << <pblocknum, threadnum >> > (mParPos, mParVel, parflag, parNumNow, phisolid, gridstart, gridend);
	else
		CollisionWithSolid_k << <pblocknum, threadnum >> > (mParPos, mParVel, parflag, parNumNow, phisolid, solidux, soliduy, soliduz, mscene, bounceVelParam, bouncePosParam);

	hipDeviceSynchronize();
}

void cspray::updateLatentHeat()
{
	updateLatentHeat_k << <pblocknum, threadnum >> >(parTemperature, parLHeat, parflag, parNumNow, meltingpoint, boilingpoint, LiquidHeatTh);
	hipDeviceSynchronize();
}

void cspray::genAirFromSolid()
{
	// 	static int *daddparnums;
	// 	int haddparnums=0;
	// 	static bool first=true;
	// 	if(first)
	// 	{
	// 		hipMalloc( (void**)&daddparnums, sizeof(int));
	// 		first=false;
	// 	}
	// 	hipMemcpy( daddparnums, &haddparnums, sizeof(int), hipMemcpyHostToDevice );
	// 
	// 	genAirFromSolid_k<<<gsblocknum, threadnum>>>( mParPos, mParVel, parflag, parsolubility, pargascontain, parmass, parTemperature, parNumNow,
	// 		mmark, phisolid, Tp, daddparnums, randfloat, randfloatcnt, mframe );
	// 
	// 	hipMemcpy( &haddparnums, daddparnums, sizeof(int), hipMemcpyDeviceToHost );
	// 	parNumNow += haddparnums;
	// 	pblocknum = max(1,(int)ceil(((float)parNumNow)/threadnum));
	// 	printf("add particle = %d\n", haddparnums );
}

void cspray::pouring()
{
	if (mframe % 4 == 0 && pourNum != 0 && pourNum + parNumNow <= parNumMax)
	{
		int tpblocknum = max(1, (int)ceil(((float)pourNum) / threadnum));
		//	printf("pournum=%d,parnumnow=%d,parnummax=%d,blocknum=%d", pourNum, parNumNow, parNumMax, tpblocknum );
		pouringwater << <tpblocknum, threadnum >> > (mParPos, mParVel, parmass, parflag, parTemperature, parLHeat, pargascontain, parNumNow,
			dpourpos, dpourvel, TYPEFLUID, pourNum, randfloat, randfloatcnt, 0, posrandparam, velrandparam, defaultLiquidT, LiquidHeatTh);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");

		parNumNow += pourNum;
		pblocknum = max(1, (int)ceil(((float)parNumNow) / threadnum));
	}
}

void cspray::pouringgas()
{
	if (mframe % 6 == 0 && pourNum != 0 && pourNum + parNumNow <= parNumMax)
	{
		int tpblocknum = max(1, (int)ceil(((float)pourNum) / threadnum));
		//	printf("pournum=%d,parnumnow=%d,parnummax=%d,blocknum=%d", pourNum, parNumNow, parNumMax, tpblocknum );
		pouringwater << <tpblocknum, threadnum >> > (mParPos, mParVel, parmass, parflag, parTemperature, parLHeat, pargascontain, parNumNow,
			dpourpos, dpourvel, TYPEAIRSOLO, pourNum, randfloat, randfloatcnt, 0, posrandparam, velrandparam, defaultLiquidT, LiquidHeatTh);
		hipDeviceSynchronize();
		getLastCudaError("Kernel execution failed");

		parNumNow += pourNum;
		pblocknum = max(1, (int)ceil(((float)parNumNow) / threadnum));
	}
}

////liquid��SPH���棬������һ���������
void cspray::liquidUpdate_SPH()
{
	calcDensPressSPH_SLCouple << <pblocknum, threadnum >> >(mParPos, pardens, parpress, parflag, parNumNow, gridstart, gridend);
	hipDeviceSynchronize();

	//debug: check the density
	if (0)
	{
		static float *hdens = new float[parNumMax];
		static char *hflag = new char[parNumMax];
		hipMemcpy(hdens, pardens, parNumNow*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(hflag, parflag, parNumNow*sizeof(char), hipMemcpyDeviceToHost);

		float densaver = 0, densmax = -1, densmin = 10000000;
		int cnt = 0;
		for (int i = 0; i<parNumNow; i++)
		{
			densaver += 1 / hdens[i];
			densmax = max(densmax, 1 / hdens[i]);
			densmin = min(densmin, 1 / hdens[i]);
			cnt++;
		}
		densaver /= cnt;
		printf("densaver, max, min=%f,%f,%f\n", densaver, densmax, densmin);
	}

	enforceForceSPH_SLCouple << <pblocknum, threadnum >> >(mParPos, mParVel, pardens, parpress, parflag, parNumNow, gridstart, gridend, viscosiySPH);
	hipDeviceSynchronize();
}

void cspray::initHeatAlphaArray()
{
	float *halpha = new float[TYPECNT];
	halpha[TYPEFLUID] = heatalphafluid;
	halpha[TYPEAIR] = heatalphaair;
	halpha[TYPEVACUUM] = heatalphavacuum;
	halpha[TYPESOLID] = heatalphasolid;
	halpha[TYPEAIRSOLO] = heatalphaair;
	halpha[TYPEBOUNDARY] = 0;

	hipMalloc((void**)&HeatAlphaArray, TYPECNT*sizeof(float));
	hipMemcpy(HeatAlphaArray, halpha, TYPECNT*sizeof(float), hipMemcpyHostToDevice);

	delete[] halpha;
}

